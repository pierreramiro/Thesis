// ING618 Algoritmos Paralelos - Prefix Sum
// Kernel code from Mark Harris - Parallel Prefix Sum (Scan) with CUDA
// Optimal in W, limited to sizes up to 2048 elements.

// Scan03: Adding proper offset on shared memory to avoid bank conflicts.

#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h> // for rand();
#include <time.h> // to use clock() functions

#define ngpu 1024
#define threadsPerBlock 512 // threads per block
#define iter 10000
#define iterCPU 100000

#define NUM_BANKS 32
#define LOG_NUM_BANKS 5
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS)

void iScan( float* output, float* input, int length);
void eScan( float* output, float* input, int length);

__global__ void eScanGPU(float *g_odata, float *g_idata, int n)
{
	__shared__ float temp[2*ngpu];// allocated on invocation
	int thid = threadIdx.x;
	int offset = 1;
	
	int ai = thid;
	int bi = thid + (n/2);
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
	int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
	temp[ai + bankOffsetA] = g_idata[ai];
	temp[bi + bankOffsetB] = g_idata[bi];

//	temp[2*thid] = g_idata[2*thid]; // load input into shared memory
//	temp[2*thid+1] = g_idata[2*thid+1];
	
	for (int d = n>>1; d > 0; d >>= 1) // build sum in place up the tree
	{
		__syncthreads();
		if (thid < d)
		{
			int ai = offset*(2*thid+1)-1;
			int bi = offset*(2*thid+2)-1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);
//			int ai = offset*(2*thid+1)-1;
//			int bi = offset*(2*thid+2)-1;
			temp[bi] += temp[ai];
		}
		offset *= 2;
	}
	if (thid==0) { temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0; }
	//	if (thid == 0) { temp[n - 1] = 0; } // clear the last element
	for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (thid < d)
		{
			int ai = offset*(2*thid+1)-1;
			int bi = offset*(2*thid+2)-1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);
//			int ai = offset*(2*thid+1)-1;
//			int bi = offset*(2*thid+2)-1;

			float t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();
	g_odata[ai] = temp[ai + bankOffsetA];
	g_odata[bi] = temp[bi + bankOffsetB];
//	g_odata[2*thid] = temp[2*thid]; // write results to device memory
//	g_odata[2*thid+1] = temp[2*thid+1];
}

int main()
{

	float *in,*outgpu,*outcpu;
	hipError_t cudaerr;
	int z;

	in  = (float *)malloc(ngpu*sizeof(float)); // input data
	outgpu = (float *)malloc(ngpu*sizeof(float)); // output data
	outcpu = (float *)malloc(ngpu*sizeof(float)); // output data


	// Fill data
	for(z=0;z<ngpu;z++)
	{
		in[z]=rand()%8; // Numbers between 0 and 7
	}
	in[0]=0;

	printf("\n %f %f %f %f %f\n",in[0],in[1],in[2],in[3],in[4]);
	// Setup timing using cudaEvent
	hipEvent_t start, stop;
	float gpu_time;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	// GPU using naive kernel (based on bs)
	printf("\nGPU eScan\n");

	int numBlocks;
    float *d_src = NULL;
    float *d_dst = NULL;

	hipMalloc((void **)(&d_src), sizeof(float) * ngpu); // Input data
	// Move padded input image from Host to Device
	cudaerr = hipMemcpy(d_src,in,sizeof(float)*ngpu,hipMemcpyHostToDevice);
	if (cudaerr!=0)	printf("ERROR copying in data to d_src (Host to Dev). CudaMalloc value=%i\n\r", cudaerr);
    hipMalloc((void **)(&d_dst), sizeof(float) * ngpu); // Output data

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(eScanGPU), hipFuncCachePreferL1);

	hipEventRecord(start);

	// Launch kernels
	numBlocks = 1; // WARNING: This must be an integer!!!, if not, add more code

	//Warmup
	eScanGPU <<<numBlocks,threadsPerBlock>>>(d_dst,d_src,ngpu);

	for(z=0;z<iter;z++)
	{
		eScanGPU <<<numBlocks,threadsPerBlock>>>(d_dst,d_src,ngpu);
	}

	hipEventRecord( stop );
	hipEventSynchronize( stop );
	hipEventElapsedTime( &gpu_time, start, stop );
	printf("GPU Time:  %fms\n\r",gpu_time/iter);
	printf("eScanGPU Speed: %f MegaOps/s\n",ngpu/(gpu_time/iter)/1000);
	cudaerr = hipMemcpy(outgpu,d_dst,sizeof(float)*ngpu,hipMemcpyDeviceToHost);
	if (cudaerr!=0)	printf("ERROR copying d_dst to outgpu (Dev to Host). CudaMalloc value=%i\n\r", cudaerr);

	clock_t startCPU;
	clock_t finishCPU;

	printf("\nCPU using eScan:\n");
	startCPU = clock();
	for(z=0;z<iterCPU;z++)
	{
		eScan(outcpu,in,ngpu);
	}
	finishCPU = clock();
	printf("CPU serial: %fms\n", (double)(finishCPU - startCPU)/1000/iterCPU);/// CLK_TCK);
	printf("eScanCPU Speed: %f MegaOps/s\n",ngpu/((double)(finishCPU - startCPU))/1000*iterCPU);
	// verify gpu vs cpu results
	for (z=0;z<ngpu;z++)
	{
		if (outgpu[z] != outcpu[z])
		{
			//error += abs(filteredImage[z] - filteredImageSerial[z]);
			printf("ERROR between CPU and GPU Scan on index: %i\n",z);
			printf("CPU: %f %f,%f %f\n", outcpu[z],outcpu[z+1],outcpu[z+2],outcpu[z+3]);
			printf("GPU: %f %f,%f %f\n", outgpu[z],outgpu[z+1],outgpu[z+2],outgpu[z+3]);
		
		}
	}


	printf("\n All DONE, press any key to end");


    hipFree(d_src);
    hipFree(d_dst);
	free(in);
	free(outcpu);
	free(outgpu);

    return 0;
}

void iScan( float* output, float* input, int length)
{

	output[0] = input[0]; 
	for(int z = 1; z < length; ++z)
	{
		output[z] = input[z] + output[z-1];
	}
}

void eScan( float* output, float* input, int length)
{

	output[0] = 0;
	for(int z = 1; z < length; ++z)
	{
		output[z] = input[z-1] + output[z-1];
	}
}
