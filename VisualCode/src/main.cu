#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdint.h>
#include <stdlib.h>
#include "ouster_reconstruction.h"
//CUDA libraries
#define threadsPerBlock 8  //
#define numBlocks (1024/threadsPerBlock) //(n_AZBLK/1024)
#include "hip/hip_runtime.h"
#include ""
#ifndef __HIPCC__  
	#define __HIPCC__
	#include <hip/device_functions.h>
#endif

/**
 * \brief Devices functions. 
 */
__device__ void mult_matrix_dev(double* A,unsigned int m,unsigned int n,double*B,unsigned int l,double*C) {
    for (unsigned int i = 0; i < l;i++) {
        for (unsigned int j = 0; j < m; j++) {
            C[j*l+i]=0;
            for (unsigned int k = 0; k < n; k++) {
                C[j * l +i] += A[j*n+k]*B[k*l + i];
            }
        }
    }
}
__device__ void rot_x_axis_dev(double* XYZ_points,double angle ){
    double temp[3], rot_matrix[9] = { 1,0,0,0,cos(angle),-sin(angle),0,sin(angle),cos(angle) };
    temp[0]=XYZ_points[0];
    temp[1]=XYZ_points[1];
    temp[2]=XYZ_points[2];
    mult_matrix_dev(rot_matrix, 3, 3, temp, 1, XYZ_points);
}

__device__ double eq_line_dev(double m,double x,double xb,double yb) {
    double y= m*(x-xb)+yb;
    return y;
}

__global__ void cudaGenerateAZBLK(double* Point_Cloud){
    double beam_altitude_angles[n_beams]= {15.379*D180_MPI,13.236*D180_MPI,11.128*D180_MPI,9.03*D180_MPI,6.941*D180_MPI,4.878*D180_MPI,2.788*D180_MPI,0.705*D180_MPI,-1.454*D180_MPI,-3.448*D180_MPI,-5.518*D180_MPI,-7.601*D180_MPI,-9.697*D180_MPI,-11.789*D180_MPI,-13.914*D180_MPI,-16.062*D180_MPI};
    double beam_azimuth_angles[n_beams] = { -1.24*D180_MPI, -1.2145*D180_MPI, -1.1889*D180_MPI, -1.1634*D180_MPI, -1.1379*D180_MPI, -1.1123*D180_MPI, -1.0868*D180_MPI, -1.0613*D180_MPI, -1.0357*D180_MPI, -1.0102*D180_MPI, -0.98467*D180_MPI, -0.95913*D180_MPI, -0.9336*D180_MPI, -0.90807*D180_MPI, -0.88253*D180_MPI, -0.857*D180_MPI };
    int thid = threadIdx.x + blockIdx.x * blockDim.x;//thid value from 0 to 1023
	//Generamos el primer azimuth 
    if (thid<n_beams*3){
        int index_nBeams=thid/3;
        int index_XYZ=thid-index_nBeams*3;
        double XYZ[3];
        //ubicamos el punto del azimut referencial en el plano XZ
        XYZ[0] = Radius_sphere* cos(beam_altitude_angles[index_nBeams] - MPI_2); //x
        XYZ[1] = 0;                                                              //y
        XYZ[2] = Radius_sphere * sin(beam_altitude_angles[index_nBeams] - MPI_2);//z
        //Realizamos la rotacion del punto con respecto al eje x debido al desfase
        rot_x_axis_dev(XYZ,beam_azimuth_angles[index_nBeams]);
        Point_Cloud[3 * index_nBeams + 0] = XYZ[0];
        Point_Cloud[3 * index_nBeams + 1] = XYZ[1];
        Point_Cloud[3 * index_nBeams + 2] = XYZ[2];
        /*Creamos los azimuts que inician en cada sector*/
        if (index_XYZ==0){
            Point_Cloud[(index_nBeams + n_AZBLK / 4 * n_beams)*3+0] = XYZ[0];
            Point_Cloud[(index_nBeams + n_AZBLK / 2 * n_beams)*3+0] = XYZ[0];
            Point_Cloud[(index_nBeams + n_AZBLK * 3 / 4 * n_beams)*3 +0] = XYZ[0];
        }else if (index_XYZ==1){
            Point_Cloud[(index_nBeams+ n_AZBLK / 4 * n_beams)*3+1] = XYZ[2];
            Point_Cloud[(index_nBeams + n_AZBLK / 2 * n_beams)*3+1] = -XYZ[1];
            Point_Cloud[(index_nBeams + n_AZBLK * 3 / 4 * n_beams)*3 +1] = -XYZ[2];
        }else{
            Point_Cloud[(index_nBeams + n_AZBLK / 4 * n_beams)*3+2] = -XYZ[1];
            Point_Cloud[(index_nBeams + n_AZBLK / 2 * n_beams)*3+2] = -XYZ[2];
            Point_Cloud[(index_nBeams + n_AZBLK * 3 / 4 * n_beams)*3 +2] = XYZ[1];
        }
    }
}

__global__ void cudaGenerateDonut(double* Point_Cloud){
    int thid = threadIdx.x + blockIdx.x * blockDim.x;//thid value from 0 to 1023
    double temp[3];
    if(thid>0){
        double rot_matrix[9] = { 1,0,0,0,cos(angle_between_azimuths*(double)thid),-sin(angle_between_azimuths*(double)thid),0,sin(angle_between_azimuths*(double)thid),cos(angle_between_azimuths*(double)thid) };
        double XYZ[3];
        for (int j = 0; j < n_beams;j++) {
            //Obtain point from referencial azimuth
            XYZ[0] = Point_Cloud[j*3 + 0];
            XYZ[1] = Point_Cloud[j*3 + 1];
            XYZ[2] = Point_Cloud[j*3 + 2];
            //rotate that point
            mult_matrix_dev (rot_matrix,3,3,XYZ,1,temp);
            //Set the new azimuth
            Point_Cloud[(thid * n_beams + j)*3 + 0] = temp[0];
            Point_Cloud[(thid * n_beams + j)*3 + 1] = temp[1];
            Point_Cloud[(thid * n_beams + j)*3 + 2] = temp[2];
        }
    }
}

__global__ void cudaGenerateSphere(double* Point_Cloud){
    int thid = threadIdx.x + blockIdx.x * blockDim.x;//thid value from 0 to 1023
    double temp[3];
    //Procedemos a rotar la Donut generada
    for (unsigned int i = 1; i < n_donuts; i++) {
        double rot_motor_matrix[9]={ cos(rot_angle*(double)i),-sin(rot_angle*(double)i),0,sin(rot_angle*(double)i),cos(rot_angle*(double)i) ,0,0,0,1 };
        //multiplicamos a todos los n_point_perdonut de la Donut referencial con la matriz de rotación respectiva
        for(int j=0;j<n_beams;j++){
            mult_matrix_dev(rot_motor_matrix, 3, 3, &Point_Cloud[(thid*n_beams+j) * 3], 1, temp);
            Point_Cloud[(i * n_points_perDonut + thid*n_beams+j) * 3 + 0] = temp[0];
            Point_Cloud[(i * n_points_perDonut + thid*n_beams+j) * 3 + 1] = temp[1];
            Point_Cloud[(i * n_points_perDonut + thid*n_beams+j) * 3 + 2] = temp[2];        
        }
    }
}



/**
 * \brief SupressOverlapCUDA. OR 
 */
__global__ void SupressOverlapCUDA(double* Point_Cloud){
    int thid = threadIdx.x + blockIdx.x * blockDim.x;
    //Declare temporary variables
    double x,y,y_temp,x1,y1,m;//xn,yn
    //Set vertical limits
    double X_L1=Point_Cloud[0],X_Ln=Point_Cloud[(n_beams-1)*3+0];
    //Array which contains lineś parameters
    double L[(n_donuts-2)*5];
    //Declare parameters from Donut_2 to Donut_n-1. Ya que para la supresión solo
    //se necesita los parametros de la Donut anterior y no necesitamos el de la
    //última Donut
    for (int i = 1; i < n_donuts-1; i++)
    {
        //Hallamos dos puntos de la recta proyectada en el plano XY
        x1=Point_Cloud[i*n_points_perDonut*3+0];
        y1=Point_Cloud[i*n_points_perDonut*3+1];
        x=Point_Cloud[i*n_points_perDonut*3+n_beams*3+0];
        y=Point_Cloud[i*n_points_perDonut*3+n_beams*3+1];
        //Calculamos la pendiente
        m=(y-y1)/(x-x1);
        //Guardamos los valores de un punto de la recta del beam_0
        L[(i-1)*5+0]=x1;
        L[(i-1)*5+1]=y1;
        //Guardamos los valores de un punto de la recta del beam_n
        L[(i-1)*5+2]=Point_Cloud[i*n_points_perDonut*3+n_beams*3-3];//xn
        L[(i-1)*5+3]=Point_Cloud[i*n_points_perDonut*3+n_beams*3-3+1];//yn
        //Guardamos el valor de la pendiente hallada
        L[(i-1)*5+4]=m;
    }
    //Supress redundant data only for Donut 2
    int i=1;
    for (unsigned int j = 0; j < n_beams; j++){
        //Calculamos si la coordenada del punto x
        x=Point_Cloud[(i*n_points_perDonut+thid*n_beams+j)*3+0];
        //Analizamos si se encuentra en la zona de supresión
        if (X_Ln<=x&&x<=X_L1){
            Point_Cloud[(i*n_points_perDonut+thid*n_beams+j)*3]=0;
            Point_Cloud[(i*n_points_perDonut+thid*n_beams+j)*3+1]=0;
            Point_Cloud[(i*n_points_perDonut+thid*n_beams+j)*3+2]=0;
            //points_deleted=points_deleted+1;
        }
    }
    //Supress redundant for the rest of the Donuts
    //Creamos variable booleana para saber la zona del plano en donde se encuentra el punto
    bool left_side;
    unsigned int offset;
    for (unsigned int i = 2; i < n_donuts; i++){
        for (unsigned int j = 0; j < n_beams; j++){
            //Hallamos las coordenadas del punto a analizar
            offset=(i*n_points_perDonut+thid*n_beams+j)*3;
            x=Point_Cloud[offset];
            y=Point_Cloud[offset+1];
            //Evaluamos si se encuentra en la zona referencial
            if (X_Ln<=x){
                if(x<=X_L1){
                    Point_Cloud[offset]=0;
                    Point_Cloud[offset+1]=0;
                    Point_Cloud[offset+2]=0;
                    continue;
                }else{
                    //Se encuentra del lado derecho
                    left_side=false;
                }
            }else{
                //Se encuentra del lazo izquierdo
                left_side=true;
            }
            //Calculamos el valor de y_temp el cual limitará la zona
            y_temp=eq_line_dev(L[(i-2)*5+4],x,L[(i-2)*5+left_side*2],L[(i-2)*5+left_side*2+1]);
            //Le colocamos un signo negativo, o no, para poder realizar un único condicional para ambos casos
            y_temp=y_temp*(1.0-2*left_side);
            y=y*(1.0-2*left_side);
            //Evaluamos la condición de supresión
            if (y>=y_temp){
                //Eliminamos los puntos
                Point_Cloud[offset]=0;
                Point_Cloud[offset+1]=0;
                Point_Cloud[offset+2]=0;
                //points_deleted=points_deleted+1;
            }
        }   
    }
    __syncthreads();
}
/**
 * \brief OneDonutFillCUDA.  
 */
__global__ void cudaODF_part1(double* Point_Cloud,unsigned int* T){
    int thid = threadIdx.x + blockIdx.x * blockDim.x;
    if(thid<n_AZBLK*(n_beams-1)){
        int index_AZBLK=thid/(double)(n_beams-1);
        int index_nBeams=(thid-index_AZBLK*(n_beams-1));
        //T[thid*3]=thid;
        //T[thid*3+1]=index_AZBLK;
        //T[thid*3+2]=index_nBeams;
        
        T[index_AZBLK*(n_beams-1)*3*2+index_nBeams*6]=thid;
        T[index_AZBLK*(n_beams-1)*3*2+index_nBeams*6+1]=index_AZBLK;
        T[index_AZBLK*(n_beams-1)*3*2+index_nBeams*6+2]=index_nBeams;
        
        T[index_AZBLK*(n_beams-1)*3*2+index_nBeams*6+3]=thid;
        T[index_AZBLK*(n_beams-1)*3*2+index_nBeams*6+4]=index_AZBLK;
        T[index_AZBLK*(n_beams-1)*3*2+index_nBeams*6+5]=index_nBeams;

    
    }else{
    //if(thid<n_AZBLK*(n_beams-1)){
        int index_AZBLK=thid/(double)(n_beams-1);
        int index_nBeams=(thid-index_AZBLK*(n_beams-1));
        unsigned int v0,v1,v2;
        //Definimos los vértices
        //Realizamos la malla triangular para la Donut referencial
        v0=index_AZBLK*n_beams+index_nBeams;
        v2=v0+1;
        v1=(v0+n_beams+1)&mask;
        T[index_AZBLK*(n_beams-1)*3*2+index_nBeams*6]=v0;
        T[index_AZBLK*(n_beams-1)*3*2+index_nBeams*6+1]=v1;
        T[index_AZBLK*(n_beams-1)*3*2+index_nBeams*6+2]=v2;
        v2=v1;
        v1=v2-1;
        T[index_AZBLK*(n_beams-1)*3*2+index_nBeams*6+3]=v0;
        T[index_AZBLK*(n_beams-1)*3*2+index_nBeams*6+4]=v1;
        T[index_AZBLK*(n_beams-1)*3*2+index_nBeams*6+5]=v2;
    }
}

__global__ void cudaODF_part2(double* Point_Cloud,unsigned int* T,unsigned int* flag_array){
    int thid = threadIdx.x + blockIdx.x * blockDim.x;
    if (thid<n_triangles_perDonut*(n_beams-1)){ //thid from  0 to 184320
        int index_Donut=thid/(double)n_triangles_perDonut+1;
        int index_triPerDonut=thid-(index_Donut-1)*n_triangles_perDonut;
        double xp,yp,zp;
        unsigned int flag=0;
        unsigned int temp_vex=(T[index_triPerDonut*3]+index_Donut*n_points_perDonut);
        xp=Point_Cloud[temp_vex*3+0];
        yp=Point_Cloud[temp_vex*3+1];
        zp=Point_Cloud[temp_vex*3+2];
            
        if ((xp!=0)||(yp!=0)||(zp!=0)){
            //analizamos el punto del vertice v1
            temp_vex=(T[index_triPerDonut*3+1]+index_Donut*n_points_perDonut);
            xp=Point_Cloud[temp_vex*3+0];
            yp=Point_Cloud[temp_vex*3+1];
            zp=Point_Cloud[temp_vex*3+2];
            if ((xp!=0)||(yp!=0)||(zp!=0)){
                //analizamos el punto del vertice v2
                temp_vex=(T[index_triPerDonut*3+2]+index_Donut*n_points_perDonut);
                xp=Point_Cloud[temp_vex*3+0];
                yp=Point_Cloud[temp_vex*3+1];
                zp=Point_Cloud[temp_vex*3+2];
                if ((xp!=0)||(yp!=0)||(zp!=0)){
                    //Si todo lo anterior se cumple, guardamos el triángulo
                    flag=1;
                }
            }
        }
        flag_array[thid-n_triangles_perDonut]=flag;
    }  
}
void cpuODF_part3 (double* Point_Cloud,unsigned int *T,unsigned int *flag_array){
    //analizamos los flags
    int count=0;
    for (unsigned int z = 0; z < n_triangles_perDonut*(n_donuts-1); z++)
    {
        if (flag_array[z]==1){
            count++;
        }
    }
    printf ("%d\n",count);
    

}

__global__ void ODF_part1(double* Point_Cloud,unsigned int* T,unsigned int* T_temp,unsigned int* count_array){
    int thid = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int v0,v1,v2;
    //Definimos los vértices
    //Realizamos la malla triangular para la Donut referencial
    for (unsigned int j = 0; j < n_beams-1; j++)
    {   
        v0=thid*n_beams+j;
        v2=v0+1;
        v1=(v0+n_beams+1)&mask;
        T[thid*(n_beams-1)*3*2+j*6]=v0;
        T[thid*(n_beams-1)*3*2+j*6+1]=v1;
        T[thid*(n_beams-1)*3*2+j*6+2]=v2;
        v2=v1;
        v1=v2-1;
        T[thid*(n_beams-1)*3*2+j*6+3]=v0;
        T[thid*(n_beams-1)*3*2+j*6+4]=v1;
        T[thid*(n_beams-1)*3*2+j*6+5]=v2;
    }
    //__syncthreads(); Not necessary
    //En base a la malla referencial hallamos las demás superficies
    double xp,yp,zp;
    unsigned int count=0,offset,temp_vex,n_triangles_perThreadandDonut=2*(n_beams-1);
    for (unsigned int i = 1; i < n_donuts; i++){
        //Analizamos cada vertice del tríangulo
        for (unsigned int j = 0; j < n_triangles_perThreadandDonut; j++){
             //Analizamos el punto del vertice v0
            offset=(thid*n_triangles_perThreadandDonut+j)*3;
            temp_vex=(T[offset]+i*n_points_perDonut);
            xp=Point_Cloud[temp_vex*3+0];
            yp=Point_Cloud[temp_vex*3+1];
            zp=Point_Cloud[temp_vex*3+2];
            if ((xp!=0)||(yp!=0)||(zp!=0)){
                //analizamos el punto del vertice v1
                temp_vex=(T[offset+1]+i*n_points_perDonut);
                xp=Point_Cloud[temp_vex*3+0];
                yp=Point_Cloud[temp_vex*3+1];
                zp=Point_Cloud[temp_vex*3+2];
                if ((xp!=0)||(yp!=0)||(zp!=0)){
                    //analizamos el punto del vertice v2
                    temp_vex=(T[offset+2]+i*n_points_perDonut);
                    xp=Point_Cloud[temp_vex*3+0];
                    yp=Point_Cloud[temp_vex*3+1];
                    zp=Point_Cloud[temp_vex*3+2];
                    if ((xp!=0)||(yp!=0)||(zp!=0)){
                        //Si todo lo anterior se cumple, guardamos el triángulo
                        T_temp[(n_donuts-1)*n_triangles_perThreadandDonut*3*thid+count*3+2]=temp_vex;
                        T_temp[(n_donuts-1)*n_triangles_perThreadandDonut*3*thid+count*3+1]=(T[offset+1]+i*n_points_perDonut);
                        T_temp[(n_donuts-1)*n_triangles_perThreadandDonut*3*thid+count*3]=(T[offset]+i*n_points_perDonut);
                        count++;
                    }
                }
            }
        }
    }
    count_array[thid]=count;
    __syncthreads();
}
#define ngpu 1024
#define NUM_BANKS 32
#define LOG_NUM_BANKS 5
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS)
__global__ void eScanGPU(unsigned int *g_odata, unsigned int *g_idata)
{
	__shared__ unsigned int temp[2*ngpu];// allocated on invocation
	int thid = threadIdx.x;
	int offset = 1,n=ngpu;
	
	int ai = thid;
	int bi = thid + (n/2);
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
	int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
	temp[ai + bankOffsetA] = g_idata[ai];
	temp[bi + bankOffsetB] = g_idata[bi];

//	temp[2*thid] = g_idata[2*thid]; // load input into shared memory
//	temp[2*thid+1] = g_idata[2*thid+1];
	
	for (int d = n>>1; d > 0; d >>= 1) // build sum in place up the tree
	{
		__syncthreads();
		if (thid < d)
		{
			int ai = offset*(2*thid+1)-1;
			int bi = offset*(2*thid+2)-1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);
//			int ai = offset*(2*thid+1)-1;
//			int bi = offset*(2*thid+2)-1;
			temp[bi] += temp[ai];
		}
		offset *= 2;
	}
	if (thid==0) { temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0; }
	//	if (thid == 0) { temp[n - 1] = 0; } // clear the last element
	for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (thid < d)
		{
			int ai = offset*(2*thid+1)-1;
			int bi = offset*(2*thid+2)-1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);
//			int ai = offset*(2*thid+1)-1;
//			int bi = offset*(2*thid+2)-1;

			unsigned int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();
	g_odata[ai] = temp[ai + bankOffsetA];
	g_odata[bi] = temp[bi + bankOffsetB];
//	g_odata[2*thid] = temp[2*thid]; // write results to device memory
//	g_odata[2*thid+1] = temp[2*thid+1];
}

__global__ void ODF_part2(unsigned int* T,unsigned int* T_temp,unsigned int* count_array,unsigned int* index_offset){
    int thid = threadIdx.x + blockIdx.x * blockDim.x;   
    unsigned int count,offset,n_triangles_perThreadandDonut=2*(n_beams-1);
    count=count_array[thid];
    //Ha este punto, cada hilo contiene una cierta cantidad de n triangulos que han de ser colocadas en el array original
    if(thid==0)
    offset=0;
    else
    offset=index_offset[thid]*3;
    //copy triangles 
    for(int z=0;z<count*3;z++){
        T[n_triangles_perDonut*3+offset+z]=T_temp[(n_donuts-1)*n_triangles_perThreadandDonut*3*thid+z];
    }
    __syncthreads();
}

/**
 * \brief Generate surface.  
 */
void Generate_surfaceGPU(double* Point_Cloud,unsigned int* T,double* Sphere_Cloud,unsigned int* T_Sphere,unsigned int *pointer_n_triangles,
                        double* Sphere_Cloud_dev,unsigned int *OneDonutMesh_dev,unsigned int *T_temp_dev,unsigned int *count_array_dev,unsigned int *index_offset_array_dev){
    /////////////////////////////////////////////////////////////////////////////////////////
    ///////////////////////////////////      CUDA        ////////////////////////////////////
    /////////////////////////////////////////////////////////////////////////////////////////
    hipError_t cudaerr;
    //1st step. GenerateSphere
    cudaGenerateAZBLK<<<numBlocks, threadsPerBlock >>> (Sphere_Cloud_dev);
    cudaGenerateDonut<<<numBlocks, threadsPerBlock >>> (Sphere_Cloud_dev);
    cudaGenerateSphere<<<numBlocks, threadsPerBlock >>> (Sphere_Cloud_dev);
    //2nd step. Overlap removing
    SupressOverlapCUDA<<<numBlocks, threadsPerBlock >>> (Sphere_Cloud_dev);
    cudaerr=hipMemcpy(Sphere_Cloud, Sphere_Cloud_dev, sizeof(double) *n_total_points * 3, hipMemcpyDeviceToHost);
    if (cudaerr != 0)	printf("ERROR copying to SphereCloud. CudaMalloc value=%i\n\r",cudaerr);
    //6th step. last Fill
    TwoandTri_Donut_Fill(Sphere_Cloud,&T_Sphere[OneDonutFill_triangles*3],&T_Sphere[(OneDonutFill_triangles+TwoDonutFill_triangles)*3],&T_Sphere[(OneDonutFill_triangles+TwoDonutFill_triangles+TriDonutFill_triangles)*3]);
    //3rd step. First part of the ODF
    ODF_part1<<<numBlocks, threadsPerBlock >>> (Sphere_Cloud_dev,OneDonutMesh_dev,T_temp_dev,count_array_dev);
    //4th step. eScan GPU
    eScanGPU<<<1, 512 >>> (index_offset_array_dev,count_array_dev);//try with CPU. Try with masking. Chapter: toolkit pin-memory. Optimized method. Chapter: asynch transfer
    /*
    unsigned int count_array[1024],index_offset_array[1024];
    cudaerr=hipMemcpy(count_array, count_array_dev, sizeof(unsigned int) *1024, hipMemcpyDeviceToHost);
    if (cudaerr != 0)	printf("ERROR copying to count_array. CudaMalloc value=%i\n\r",cudaerr);
    index_offset_array[0]=0;
    for (int z=1;z<1024;z++){
        index_offset_array[z]=index_offset_array[z-1]+count_array[z-1];
    }
    cudaerr=hipMemcpy(index_offset_array_dev, index_offset_array, sizeof(unsigned int) *1024, hipMemcpyHostToDevice);
    if (cudaerr != 0)	printf("ERROR copying to index_offset_array_dev. CudaMalloc value=%i\n\r",cudaerr);
    */
     
    
    //5th step. last part ODF
    ODF_part2<<<numBlocks, threadsPerBlock >>> (OneDonutMesh_dev,T_temp_dev,count_array_dev,index_offset_array_dev);
    cudaerr=hipMemcpy(T_Sphere,OneDonutMesh_dev, sizeof(unsigned int) *OneDonutFill_triangles * 3, hipMemcpyDeviceToHost);
    if (cudaerr != 0)	printf("ERROR copying to T_Sphere. CudaMalloc value=%i\n\r",cudaerr);
	/////////////////////////////////////////////////////////////////////////////////////////
    ///////////////////////////////////      CPU        ////////////////////////////////////
    /////////////////////////////////////////////////////////////////////////////////////////
    hipDeviceSynchronize();
    unsigned int temp_vex,n_triangles=0;
    double xp,yp,zp;
    for (unsigned int i = 0; i < n_total_triangles; i++){
        //Analizamos el punto del vertice v0
        temp_vex=T_Sphere[i*3];
        xp=Point_Cloud[temp_vex*3+0];
        yp=Point_Cloud[temp_vex*3+1];
        zp=Point_Cloud[temp_vex*3+2];
        if ((xp!=0)&&(yp!=0)&&(zp!=0)){
            //analizamos el punto del vertice v1
            temp_vex=T_Sphere[i*3+1];
            xp=Point_Cloud[temp_vex*3+0];
            yp=Point_Cloud[temp_vex*3+1];
            zp=Point_Cloud[temp_vex*3+2];
            if ((xp!=0)&&(yp!=0)&&(zp!=0)){
                //analizamos el punto del vertice v2
                temp_vex=T_Sphere[i*3+2];
                xp=Point_Cloud[temp_vex*3+0];
                yp=Point_Cloud[temp_vex*3+1];
                zp=Point_Cloud[temp_vex*3+2];
                if ((xp!=0)&&(yp!=0)&&(zp!=0)){
                    //Si todo lo anterior se cumple, guardamos el triángulo
                    T[n_triangles*3+2]=temp_vex;
                    T[n_triangles*3+1]=T_Sphere[i*3+1];
                    T[n_triangles*3]=T_Sphere[i*3];
                    n_triangles++;
                }
            }
        }
    }
    pointer_n_triangles[0]=n_triangles; 
}


/******************************************************************/
/*************************       MAIN     *************************/
/******************************************************************/
#define TESTING 1
bool malloc_already=false;
int main()
{
#if TESTING == 0
    #define iter 1000.0
    /*Allocate memory*/
    double* Point_Cloud,*Sphere_Cloud_cpu;
    Point_Cloud = (double*)malloc(n_total_points * 3 *sizeof(double));
    Sphere_Cloud_cpu = (double*)malloc(n_total_points * 3 *sizeof(double));
    //Leemos del csv los datos reales
    FILE* archivo;
    archivo = fopen("../files/MinaData.csv", "r");
    char buffer[200];
    char* token;
    //Saltamos la primera línea
    fgets(buffer,sizeof(buffer),archivo);
    for (unsigned int i = 0; i < n_total_points; i++){
        fgets(buffer,sizeof(buffer),archivo);
        token = strtok(buffer,",");
        Point_Cloud[i*3+0]=atof(token);
        token = strtok(NULL,",");
        Point_Cloud[i*3+1]=atof(token);
        token = strtok(NULL,",\n");
        Point_Cloud[i*3+2]=atof(token);
    }    
    fclose(archivo);    
    
    /*****************************************************/
    /********************       CPU     ******************/
    /*****************************************************/
    unsigned int *T_cpu,*T_Sphere_cpu,n_triangles_real_data_cpu;
    T_cpu=(unsigned int*)malloc(n_total_triangles * 3 *sizeof(unsigned int));
    T_Sphere_cpu=(unsigned int*)malloc(n_total_triangles * 3 *sizeof(unsigned int));
    clock_t startCPU;
	clock_t finishCPU;
    printf ("/********************solo CPU*********************/:\n");
	startCPU = clock();
    for (int i=0;i<iter;i++){
        Generate_surface(Point_Cloud,T_cpu,Sphere_Cloud_cpu,T_Sphere_cpu,&n_triangles_real_data_cpu);
    }
	finishCPU = clock();
	printf("numero de triangulos: %d\n",n_triangles_real_data_cpu);
    printf("CPU: %fms\n", ((double)(finishCPU - startCPU))*1000/(double)CLOCKS_PER_SEC/iter);
	
    /*****************************************************/
    /********************       GPU     ******************/
    /*****************************************************/
    unsigned int *T_gpu,*T_Sphere_gpu,n_triangles_real_data_gpu;
    T_gpu=(unsigned int*)malloc(n_total_triangles * 3 *sizeof(unsigned int));
    T_Sphere_gpu=(unsigned int*)malloc(n_total_triangles * 3 *sizeof(unsigned int));
    double *Sphere_Cloud_gpu;
    Sphere_Cloud_gpu = (double*)malloc(n_total_points * 3 *sizeof(double));
    

    double* Sphere_Cloud_dev;
    unsigned int *OneDonutMesh_dev,*T_temp_dev,*count_array_dev,*index_offset_array_dev;
    hipMalloc((void**)(&Sphere_Cloud_dev), sizeof(double) * n_total_points * 3);
    hipMalloc((void**)(&OneDonutMesh_dev), sizeof(unsigned int) * OneDonutFill_triangles * 3);
    hipMalloc((void**)(&T_temp_dev), sizeof(unsigned int) * n_triangles_perDonut* (n_donuts-1) * 3);
    hipMalloc((void**)(&count_array_dev), sizeof(unsigned int) * 1024);
    hipMalloc((void**)(&index_offset_array_dev), sizeof(unsigned int) * 1024);

    clock_t startGPU;
	clock_t finishGPU;
    printf ("/********************CPU y GPU*********************/:\n");
	startGPU = clock();
    for (int i=0;i<iter;i++){
        Generate_surfaceGPU(Point_Cloud,T_gpu,Sphere_Cloud_gpu,T_Sphere_gpu,&n_triangles_real_data_gpu,
                            Sphere_Cloud_dev,OneDonutMesh_dev,T_temp_dev,count_array_dev,index_offset_array_dev);
    }
	finishGPU = clock();
	printf("numero de triangulos: %d\n",n_triangles_real_data_gpu);
    printf("GPU: %fms\n", ((double)(finishGPU - startGPU))*1000/(double)CLOCKS_PER_SEC/iter);
    hipFree(Sphere_Cloud_dev);
    hipFree(OneDonutMesh_dev);
    hipFree(T_temp_dev);
    hipFree(count_array_dev);
    hipFree(index_offset_array_dev); 
    //creamos archivo para ver results
    archivo = fopen("../files/CUDAMesh.csv", "w+");
    fprintf(archivo, "V1, V2, V3\n");
    for (unsigned int i=0; i < n_triangles_real_data_gpu; i++) {
        fprintf(archivo,"%d, %d, %d\n", T_gpu[i*3+0], T_gpu[i * 3 + 1], T_gpu[i * 3 + 2]);
    }
    fclose(archivo);
    //------------------------------------------
	//----------Generate the DXF file-----------
	//------------------------------------------
	//Open the DXF file
    archivo = fopen("../files/CUDAMinaSurface.dxf", "w");
    //assert(archivo);
    //header
    fprintf(archivo, "0\nSECTION\n2\nENTITIES\n0\n");
	float x0,y0,z0,x1,y1,z1,x2,y2,z2;
	for (int i = 0; i < n_triangles_real_data_gpu; i++)
    {
        // get the coordinates of each point from the triangle
        x0 = Point_Cloud[T_gpu[i* 3+0]*3 + 0];
        y0 = Point_Cloud[T_gpu[i* 3+0]*3 + 1];
        z0 = Point_Cloud[T_gpu[i* 3+0]*3 + 2];
        
        x1 = Point_Cloud[T_gpu[i* 3+1]*3 + 0];
        y1 = Point_Cloud[T_gpu[i* 3+1]*3 + 1];
        z1 = Point_Cloud[T_gpu[i* 3+1]*3 + 2];
        
        x2 = Point_Cloud[T_gpu[i* 3+2]*3 + 0];
        y2 = Point_Cloud[T_gpu[i* 3+2]*3 + 1];
        z2 = Point_Cloud[T_gpu[i* 3+2]*3 + 2];
        //create new 3DFACE element
        fprintf(archivo, "3DFACE\n8\n1\n");
        fprintf(archivo, " 62\n %d\n", 142);//corresponding color of the autocad pallete
        fprintf(archivo, "10\n %.4f\n 20\n %.4f\n 30\n %.4f\n", x0, y0, z0);
        fprintf(archivo, "11\n %.4f\n 21\n %.4f\n 31\n %.4f\n", x1, y1, z1);
        fprintf(archivo, "12\n %.4f\n 22\n %.4f\n 32\n %.4f\n", x2, y2, z2);
        fprintf(archivo, "13\n %.4f\n 23\n %.4f\n 33\n %.4f\n", x2, y2, z2);
        fprintf(archivo, "0\n");
    }
    fprintf(archivo, "ENDSEC\n 0\nEOF\n");
    fclose(archivo);
    free (Point_Cloud);
    free (Sphere_Cloud_cpu);
    free (T_cpu);
    free (T_Sphere_cpu);
    return 0;
#elif TESTING == 1
    #define iter 100.0
    /*************************************************************************************/
    /***********************************    CPU     **************************************/
    /*************************************************************************************/
    double* Point_Cloud;
    Point_Cloud = (double*)malloc(n_total_points * 3 *sizeof(double));
    unsigned int *T;
    T=(unsigned int*)malloc(n_total_triangles * 3 *sizeof(unsigned int));

    clock_t start,stop;
    double timeCPU=0;
    printf ("/********************  CPU  *********************/:\n");
    start=clock();
    for (int z=0;z<iter;z++)
    Generate_sphere(Point_Cloud);
    stop=clock();
    timeCPU+=((double)(stop - start))*1000.0/(double)CLOCKS_PER_SEC/iter;
    printf("GS time: %fms\n", ((double)(stop - start))*1000.0/(double)CLOCKS_PER_SEC/iter);
    
    start=clock();
    for (int z=0;z<iter;z++)
    Supress_redundant_data(Point_Cloud);
    stop=clock();
    timeCPU+=((double)(stop - start))*1000.0/(double)CLOCKS_PER_SEC/iter;
    printf("OR time: %fms\n", ((double)(stop - start))*1000.0/(double)CLOCKS_PER_SEC/iter);
    
    start=clock();
    for (int z=0;z<iter;z++){
        //One_Donut_Fill(Point_Cloud,T);
        unsigned int v0,v1,v2;
        //Definimos los vértices
        //Realizamos la malla triangular para la Donut referencial
        for (unsigned int j = 0; j < n_AZBLK; j++){
            for (unsigned int k = 0; k < n_beams-1; k++)
            {   
                v0=j*n_beams+k;
                v2=v0+1;
                v1=(v0+n_beams+1)&mask;
                T[j*(n_beams-1)*3*2+k*6]=v0;
                T[j*(n_beams-1)*3*2+k*6+1]=v1;
                T[j*(n_beams-1)*3*2+k*6+2]=v2;
                v2=v1;
                v1=v2-1;
                T[j*(n_beams-1)*3*2+k*6+3]=v0;
                T[j*(n_beams-1)*3*2+k*6+4]=v1;
                T[j*(n_beams-1)*3*2+k*6+5]=v2;
            }
        }
    }
    stop=clock();
    timeCPU+=((double)(stop - start))*1000.0/(double)CLOCKS_PER_SEC/iter;
    printf("ODF_pt1 time: %fms\n", ((double)(stop - start))*1000.0/(double)CLOCKS_PER_SEC/iter);
    
    start=clock();
    for (int z=0;z<iter;z++){
        //En base a la malla referencial hallamos las demás superficies
        double xp,yp,zp;
        unsigned int temp_vex,count=0;
        for (unsigned int i = 1; i < n_donuts; i++){
            //Analizamos cada vertice del tríangulo
            for (unsigned int j = 0; j < n_triangles_perDonut; j++){
                //Analizamos el punto del vertice v0
                temp_vex=(T[j*3]+i*n_points_perDonut);
                xp=Point_Cloud[temp_vex*3+0];
                yp=Point_Cloud[temp_vex*3+1];
                zp=Point_Cloud[temp_vex*3+2];
                if ((xp!=0)||(yp!=0)||(zp!=0)){
                    //analizamos el punto del vertice v1
                    temp_vex=(T[j*3+1]+i*n_points_perDonut);
                    xp=Point_Cloud[temp_vex*3+0];
                    yp=Point_Cloud[temp_vex*3+1];
                    zp=Point_Cloud[temp_vex*3+2];
                    if ((xp!=0)||(yp!=0)||(zp!=0)){
                        //analizamos el punto del vertice v2
                        temp_vex=(T[j*3+2]+i*n_points_perDonut);
                        xp=Point_Cloud[temp_vex*3+0];
                        yp=Point_Cloud[temp_vex*3+1];
                        zp=Point_Cloud[temp_vex*3+2];
                        if ((xp!=0)||(yp!=0)||(zp!=0)){
                            //Si todo lo anterior se cumple, guardamos el triángulo
                            T[n_triangles_perDonut*3+count*3+2]=temp_vex;
                            T[n_triangles_perDonut*3+count*3+1]=(T[j*3+1]+i*n_points_perDonut);
                            T[n_triangles_perDonut*3+count*3]=(T[j*3]+i*n_points_perDonut);
                            count++;
                        }
                    }
                }
            }
        }
    }
    stop=clock();
    timeCPU+=((double)(stop - start))*1000.0/(double)CLOCKS_PER_SEC/iter;
    printf("ODF_pt2 time: %fms\n", ((double)(stop - start))*1000.0/(double)CLOCKS_PER_SEC/iter);
    printf("total time to compare: %fms\n",timeCPU);
    
    start=clock();
    for (int z=0;z<iter;z++)
    TwoandTri_Donut_Fill(Point_Cloud,&T[OneDonutFill_triangles*3],&T[(OneDonutFill_triangles+TwoDonutFill_triangles)*3],&T[(OneDonutFill_triangles+TwoDonutFill_triangles+TriDonutFill_triangles)*3]);    
    stop=clock();
    timeCPU+=((double)(stop - start))*1000.0/(double)CLOCKS_PER_SEC/iter;
    printf("LastFill time: %fms\n", ((double)(stop - start))*1000.0/(double)CLOCKS_PER_SEC/iter);
        
    free(Point_Cloud);
    free(T);
    /*************************************************************************************/
    /***********************************    GPU     **************************************/
    /*************************************************************************************/    
    double* Point_Cloud_dev,*Point_Cloud_gpu;
    hipMalloc((void**)(&Point_Cloud_dev), sizeof(double) * n_total_points * 3);
    Point_Cloud_gpu = (double*)malloc(n_total_points * 3 *sizeof(double));

    hipError_t cudaerr;
    hipEvent_t start_gpu, stop_gpu;
    float timeGPU,totalGPU=0;
    hipEventCreate(&start_gpu);
    hipEventCreate(&stop_gpu);

    printf ("/*********************  GPU  ********************/\n");
    hipEventRecord(start_gpu);
    for (int z = 0; z < iter; z++){
        //GenerateSphereCUDA<<<4, 256 >>> (Point_Cloud_dev);
        cudaGenerateAZBLK<<<numBlocks, threadsPerBlock >>> (Point_Cloud_dev);
        cudaGenerateDonut<<<numBlocks, threadsPerBlock >>> (Point_Cloud_dev);
        cudaGenerateSphere<<<numBlocks, threadsPerBlock >>> (Point_Cloud_dev);
    }
    hipEventRecord(stop_gpu);
    hipEventSynchronize(stop_gpu);
	hipEventElapsedTime(&timeGPU, start_gpu, stop_gpu);
    printf("GS time:  %fms\n\r", timeGPU / iter);
    totalGPU+=timeGPU/iter;
    
    hipEventRecord(start_gpu);
    for (int z = 0; z < iter; z++)
    SupressOverlapCUDA<<<numBlocks, threadsPerBlock  >>> (Point_Cloud_dev);
    cudaerr=hipMemcpy(Point_Cloud_gpu, Point_Cloud_dev, sizeof(double) *n_total_points * 3, hipMemcpyDeviceToHost);
    if (cudaerr != 0)	printf("ERROR copying to Point_Cloud_gpu. CudaMalloc value=%i\n\r",cudaerr);
    hipEventRecord(stop_gpu);
    hipEventSynchronize(stop_gpu);
	hipEventElapsedTime(&timeGPU, start_gpu, stop_gpu);
    printf("OR time:  %fms\n\r", timeGPU / iter);
    totalGPU+=timeGPU/iter;

    unsigned int *OneMesh_dev;
    unsigned int *flag_array_dev,*index_offset_array_dev;
    hipMalloc((void**)(&OneMesh_dev), sizeof(unsigned int) * n_triangles_perDonut * 3);
    hipMalloc((void**)(&flag_array_dev), sizeof(unsigned int) * n_triangles_perDonut*(n_donuts-1));
    hipMalloc((void**)(&index_offset_array_dev), sizeof(unsigned int) * 1024);
    unsigned int *T_gpu=(unsigned int*)malloc(n_total_triangles * 3 *sizeof(unsigned int));

    unsigned int *flag_array=(unsigned int*)malloc(n_triangles_perDonut*(n_donuts-1) *sizeof(unsigned int));
    //3rd step. First part of the ODF
    hipEventRecord(start_gpu);
    for (int z = 0; z < iter; z++){
        int temp_blok=64;
        cudaODF_part1<<<n_AZBLK*(n_beams-1)/temp_blok, temp_blok >>> (Point_Cloud_dev,OneMesh_dev);
        temp_blok=128;
        cudaODF_part2<<<n_triangles_perDonut*(n_donuts-1)/temp_blok, temp_blok >>> (Point_Cloud_dev,OneMesh_dev,flag_array_dev);
        hipDeviceSynchronize();
        cudaerr=hipMemcpy(T_gpu,OneMesh_dev, sizeof(unsigned int) *n_triangles_perDonut *3, hipMemcpyDeviceToHost);
        if (cudaerr != 0)	printf("ERROR copying to OneMesh, index %d. CudaMalloc value=%i\n\r",z,cudaerr);
        cudaerr=hipMemcpy(flag_array,flag_array_dev, sizeof(unsigned int) *n_triangles_perDonut*(n_donuts-1) , hipMemcpyDeviceToHost);
        if (cudaerr != 0)	printf("ERROR copying to flag_array, index %d. CudaMalloc value=%i\n\r",z,cudaerr);
        
        cpuODF_part3(Point_Cloud_gpu,T_gpu,flag_array);
        
        ////4th step. eScan GPU
        //eScanGPU<<<1, 1024/2 >>> (index_offset_array_dev,count_array_dev);
        ////5th step. last part ODF
        //ODF_part2<<<numBlocks, threadsPerBlock >>> (OneMesh_dev,OneMesh_temp_dev,count_array_dev,index_offset_array_dev);
        //cudaerr=hipMemcpy(T_gpu,OneMesh_dev, sizeof(unsigned int) *OneDonutFill_triangles * 3, hipMemcpyDeviceToHost);
    }
    hipEventRecord(stop_gpu);
    hipEventSynchronize(stop_gpu);
	hipEventElapsedTime(&timeGPU, start_gpu, stop_gpu);
    printf("ODF time:  %fms\n\r", timeGPU / iter);
    totalGPU+=timeGPU/iter;

    TwoandTri_Donut_Fill(Point_Cloud_gpu,&T_gpu[OneDonutFill_triangles*3],&T_gpu[(OneDonutFill_triangles+TwoDonutFill_triangles)*3],&T_gpu[(OneDonutFill_triangles+TwoDonutFill_triangles+TriDonutFill_triangles)*3]);    
    printf("total time to compare: %fms\n",totalGPU);

    FILE* archivo;
    archivo = fopen("../testfiles/CUDASphere_cloud.csv", "w+");
    fprintf(archivo, "X, Y, Z\n");
    for (unsigned int i=0; i < n_total_points; i++) {
        fprintf(archivo,"%.4f, %.4f, %.4f\n", Point_Cloud_gpu[i*3+0], Point_Cloud_gpu[i * 3 + 1], Point_Cloud_gpu[i * 3 + 2]);
    }
    fclose(archivo);
    archivo = fopen("../testfiles/CUDAOneMesh.csv", "w+");
    fprintf(archivo, "V1, V2, V3\n");
    for (unsigned int i=0; i < n_total_triangles; i++) {
        fprintf(archivo,"%d, %d, %d\n", T_gpu[i*3+0], T_gpu[i * 3 + 1], T_gpu[i * 3 + 2]);
    }
    fclose(archivo);
    printf("fin\n");
    return;

    //Finalmente, liberamos el resto de memoria
    hipFree(Point_Cloud_dev);
    hipFree(OneMesh_dev);
    hipFree(flag_array_dev);
    hipFree(index_offset_array_dev);
    free(Point_Cloud_gpu);
    free(T_gpu); 
    return 0;
#else
    #define iter 100
    /*************************************************************************************/
    /***********************************    GPU     **************************************/
    /*************************************************************************************/    
    double* Point_Cloud_dev,*Point_Cloud_gpu;
    hipMalloc((void**)(&Point_Cloud_dev), sizeof(double) * n_total_points * 3);
    Point_Cloud_gpu = (double*)malloc(n_total_points * 3 *sizeof(double));

    hipError_t cudaerr;
    hipEvent_t start_gpu, stop_gpu;
    float timeGPU,totalGPU=0;
    hipEventCreate(&start_gpu);
    hipEventCreate(&stop_gpu);

    printf ("/*********************CPU y GPU********************/\n");
    hipEventRecord(start_gpu);
    for (int z = 0; z < iter; z++){
        cudaGenerateAZBLK<<<numBlocks, threadsPerBlock >>> (Point_Cloud_dev);
        hipDeviceSynchronize();
        cudaGenerateDonut<<<numBlocks, threadsPerBlock >>> (Point_Cloud_dev);
        hipDeviceSynchronize();
        cudaGenerateSphere<<<numBlocks, threadsPerBlock >>> (Point_Cloud_dev);
        hipDeviceSynchronize();
    }
    cudaerr=hipMemcpy(Point_Cloud_gpu, Point_Cloud_dev, sizeof(double) *n_total_points * 3, hipMemcpyDeviceToHost);
    if (cudaerr != 0)	printf("ERROR copying to Point_Cloud_gpu. CudaMalloc value=%i\n\r",cudaerr);
    hipEventRecord(stop_gpu);
    hipEventSynchronize(stop_gpu);
	hipEventElapsedTime(&timeGPU, start_gpu, stop_gpu);
    printf("GS time:  %fms\n\r", timeGPU / iter);
    totalGPU+=timeGPU/iter;

    FILE* archivo;
    archivo = fopen("../testfiles/CUDASphere_cloud.csv", "w+");
    fprintf(archivo, "X, Y, Z\n");
    for (unsigned int i=0; i < n_total_points; i++) {
        fprintf(archivo,"%.4f, %.4f, %.4f\n", Point_Cloud_gpu[i*3+0], Point_Cloud_gpu[i * 3 + 1], Point_Cloud_gpu[i * 3 + 2]);
    }
    fclose(archivo);
    return;
#endif
}
