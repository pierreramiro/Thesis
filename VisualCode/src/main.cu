#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdint.h>
#include <stdlib.h>
#include "ouster_reconstruction.h"
//CUDA libraries
#define threadsPerBlock 16  //
#define numBlocks (1024/threadsPerBlock) //(n_AZBLK/1024)
#include "hip/hip_runtime.h"
#include ""
#ifndef __HIPCC__  
	#define __HIPCC__
	#include <hip/device_functions.h>
#endif

/**
 * \brief Devices functions. 
 */
__device__ void mult_matrix_dev(double* A,unsigned int m,unsigned int n,double*B,unsigned int l,double*C) {
    for (unsigned int i = 0; i < l;i++) {
        for (unsigned int j = 0; j < m; j++) {
            C[j*l+i]=0;
            for (unsigned int k = 0; k < n; k++) {
                C[j * l +i] += A[j*n+k]*B[k*l + i];
            }
        }
    }
}
__device__ void rot_x_axis_dev(double* XYZ_points,double angle ){
    double temp[3], rot_matrix[9] = { 1,0,0,0,cos(angle),-sin(angle),0,sin(angle),cos(angle) };
    temp[0]=XYZ_points[0];
    temp[1]=XYZ_points[1];
    temp[2]=XYZ_points[2];
    mult_matrix_dev(rot_matrix, 3, 3, temp, 1, XYZ_points);
}

__device__ double eq_line_dev(double m,double x,double xb,double yb) {
    double y= m*(x-xb)+yb;
    return y;
}
/**
 * \brief GenerateSphereCUDA. genera esfera 
 */
__global__ void GenerateSphereCUDA(double* Point_Cloud){
    double beam_altitude_angles[n_beams]= {15.379*D180_MPI,13.236*D180_MPI,11.128*D180_MPI,9.03*D180_MPI,6.941*D180_MPI,4.878*D180_MPI,2.788*D180_MPI,0.705*D180_MPI,-1.454*D180_MPI,-3.448*D180_MPI,-5.518*D180_MPI,-7.601*D180_MPI,-9.697*D180_MPI,-11.789*D180_MPI,-13.914*D180_MPI,-16.062*D180_MPI};
    double beam_azimuth_angles[n_beams] = { -1.24*D180_MPI, -1.2145*D180_MPI, -1.1889*D180_MPI, -1.1634*D180_MPI, -1.1379*D180_MPI, -1.1123*D180_MPI, -1.0868*D180_MPI, -1.0613*D180_MPI, -1.0357*D180_MPI, -1.0102*D180_MPI, -0.98467*D180_MPI, -0.95913*D180_MPI, -0.9336*D180_MPI, -0.90807*D180_MPI, -0.88253*D180_MPI, -0.857*D180_MPI };
    //Obtenemos el ID del thread
    int thid = threadIdx.x + blockIdx.x * blockDim.x;//thid value from 0 to 1023
	//Generamos el primer azimuth 
    if (thid<n_beams){
        //ubicamos el punto del azimut referencial en el plano XZ
        Point_Cloud[3 * thid + 0] = Radius_sphere* cos(beam_altitude_angles[thid] - MPI_2); //x
        Point_Cloud[3 * thid + 1] = 0;                                         //y
        Point_Cloud[3 * thid + 2] = Radius_sphere * sin(beam_altitude_angles[thid] - MPI_2);//z
        //Realizamos la rotacion del punto con respecto al eje x debido al desfase
        rot_x_axis_dev(&Point_Cloud[3*thid],beam_azimuth_angles[thid]);
        /*Creamos los azimuts que inician en cada sector*/
        /*mirror points from quarter Donut*/
        Point_Cloud[(thid + n_AZBLK / 4 * n_beams)*3+0] = Point_Cloud[3 * thid + 0];
        Point_Cloud[(thid+ n_AZBLK / 4 * n_beams)*3+1] = Point_Cloud[3 * thid + 2];
        Point_Cloud[(thid + n_AZBLK / 4 * n_beams)*3+2] = -Point_Cloud[3 * thid + 1];
        /*mirror points from midle Donut*/
        Point_Cloud[(thid + n_AZBLK / 2 * n_beams)*3+0] = Point_Cloud[3 * thid + 0];
        Point_Cloud[(thid + n_AZBLK / 2 * n_beams)*3+1] = -Point_Cloud[3 * thid + 1];
        Point_Cloud[(thid + n_AZBLK / 2 * n_beams)*3+2] = -Point_Cloud[3 * thid + 2];
        /*mirror points from 3 quater Donut*/
        Point_Cloud[(thid + n_AZBLK * 3 / 4 * n_beams)*3 +0] = Point_Cloud[3 * thid + 0];
        Point_Cloud[(thid + n_AZBLK * 3 / 4 * n_beams)*3 +1] = -Point_Cloud[3 * thid + 2];
        Point_Cloud[(thid + n_AZBLK * 3 / 4 * n_beams)*3 +2] = Point_Cloud[3 * thid + 1];
    }
    __syncthreads();
    /*Definimos la matrix de rotación para los azimuth*/
    /*Procedemos a realizar el barrido de cada sector para obtener la donnut referencial*/
    double temp[3];
    if(thid>0){
        double rot_matrix[9] = { 1,0,0,0,cos(angle_between_azimuths*(double)thid),-sin(angle_between_azimuths*(double)thid),0,sin(angle_between_azimuths*(double)thid),cos(angle_between_azimuths*(double)thid) };
        double XYZ[3];
        for (int j = 0; j < n_beams;j++) {
            //Obtain point from referencial azimuth
            XYZ[0] = Point_Cloud[j*3 + 0];
            XYZ[1] = Point_Cloud[j*3 + 1];
            XYZ[2] = Point_Cloud[j*3 + 2];
            //rotate that point
            mult_matrix_dev (rot_matrix,3,3,XYZ,1,temp);
            //Set the new azimuth
            Point_Cloud[(thid * n_beams + j)*3 + 0] = temp[0];
            Point_Cloud[(thid * n_beams + j)*3 + 1] = temp[1];
            Point_Cloud[(thid * n_beams + j)*3 + 2] = temp[2];
        }
    }
    __syncthreads();
    //Procedemos a rotar la Donut generada
    for (unsigned int i = 1; i < n_donuts; i++) {
        double rot_motor_matrix[9]={ cos(rot_angle*(double)i),-sin(rot_angle*(double)i),0,sin(rot_angle*(double)i),cos(rot_angle*(double)i) ,0,0,0,1 };
        //multiplicamos a todos los n_point_perdonut de la Donut referencial con la matriz de rotación respectiva
        for(int j=0;j<n_beams;j++){
            mult_matrix_dev(rot_motor_matrix, 3, 3, &Point_Cloud[(thid*n_beams+j) * 3], 1, temp);
            Point_Cloud[(i * n_points_perDonut + thid*n_beams+j) * 3 + 0] = temp[0];
            Point_Cloud[(i * n_points_perDonut + thid*n_beams+j) * 3 + 1] = temp[1];
            Point_Cloud[(i * n_points_perDonut + thid*n_beams+j) * 3 + 2] = temp[2];        
        }
    }
    __syncthreads();
}
/**
 * \brief SupressOverlapCUDA. OR 
 */
__global__ void SupressOverlapCUDA(double* Point_Cloud){
    int thid = threadIdx.x + blockIdx.x * blockDim.x;
    //Declare temporary variables
    double x,y,y_temp,x1,y1,m;//xn,yn
    //Set vertical limits
    double X_L1=Point_Cloud[0],X_Ln=Point_Cloud[(n_beams-1)*3+0];
    //Array which contains lineś parameters
    double L[(n_donuts-2)*5];
    //Declare parameters from Donut_2 to Donut_n-1. Ya que para la supresión solo
    //se necesita los parametros de la Donut anterior y no necesitamos el de la
    //última Donut
    for (int i = 1; i < n_donuts-1; i++)
    {
        //Hallamos dos puntos de la recta proyectada en el plano XY
        x1=Point_Cloud[i*n_points_perDonut*3+0];
        y1=Point_Cloud[i*n_points_perDonut*3+1];
        x=Point_Cloud[i*n_points_perDonut*3+n_beams*3+0];
        y=Point_Cloud[i*n_points_perDonut*3+n_beams*3+1];
        //Calculamos la pendiente
        m=(y-y1)/(x-x1);
        //Guardamos los valores de un punto de la recta del beam_0
        L[(i-1)*5+0]=x1;
        L[(i-1)*5+1]=y1;
        //Guardamos los valores de un punto de la recta del beam_n
        L[(i-1)*5+2]=Point_Cloud[i*n_points_perDonut*3+n_beams*3-3];//xn
        L[(i-1)*5+3]=Point_Cloud[i*n_points_perDonut*3+n_beams*3-3+1];//yn
        //Guardamos el valor de la pendiente hallada
        L[(i-1)*5+4]=m;
    }
    //Supress redundant data only for Donut 2
    int i=1;
    for (unsigned int j = 0; j < n_beams; j++){
        //Calculamos si la coordenada del punto x
        x=Point_Cloud[(i*n_points_perDonut+thid*n_beams+j)*3+0];
        //Analizamos si se encuentra en la zona de supresión
        if (X_Ln<=x&&x<=X_L1){
            Point_Cloud[(i*n_points_perDonut+thid*n_beams+j)*3]=0;
            Point_Cloud[(i*n_points_perDonut+thid*n_beams+j)*3+1]=0;
            Point_Cloud[(i*n_points_perDonut+thid*n_beams+j)*3+2]=0;
            //points_deleted=points_deleted+1;
        }
    }
    //Supress redundant for the rest of the Donuts
    //Creamos variable booleana para saber la zona del plano en donde se encuentra el punto
    bool left_side;
    unsigned int offset;
    for (unsigned int i = 2; i < n_donuts; i++){
        for (unsigned int j = 0; j < n_beams; j++){
            //Hallamos las coordenadas del punto a analizar
            offset=(i*n_points_perDonut+thid*n_beams+j)*3;
            x=Point_Cloud[offset];
            y=Point_Cloud[offset+1];
            //Evaluamos si se encuentra en la zona referencial
            if (X_Ln<=x){
                if(x<=X_L1){
                    Point_Cloud[offset]=0;
                    Point_Cloud[offset+1]=0;
                    Point_Cloud[offset+2]=0;
                    continue;
                }else{
                    //Se encuentra del lado derecho
                    left_side=false;
                }
            }else{
                //Se encuentra del lazo izquierdo
                left_side=true;
            }
            //Calculamos el valor de y_temp el cual limitará la zona
            y_temp=eq_line_dev(L[(i-2)*5+4],x,L[(i-2)*5+left_side*2],L[(i-2)*5+left_side*2+1]);
            //Le colocamos un signo negativo, o no, para poder realizar un único condicional para ambos casos
            y_temp=y_temp*(1.0-2*left_side);
            y=y*(1.0-2*left_side);
            //Evaluamos la condición de supresión
            if (y>=y_temp){
                //Eliminamos los puntos
                Point_Cloud[offset]=0;
                Point_Cloud[offset+1]=0;
                Point_Cloud[offset+2]=0;
                //points_deleted=points_deleted+1;
            }
        }   
    }
    __syncthreads();
}
/**
 * \brief OneDonutFillCUDA.  
 */
__global__ void ODF_part1(double* Point_Cloud,unsigned int* T,unsigned int* T_temp,unsigned int* count_array){
    int thid = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int v0,v1,v2;
    //Definimos los vértices
    //Realizamos la malla triangular para la Donut referencial
    for (unsigned int j = 0; j < n_beams-1; j++)
    {   
        v0=thid*n_beams+j;
        v2=v0+1;
        v1=(v0+n_beams+1)&mask;
        T[thid*(n_beams-1)*3*2+j*6]=v0;
        T[thid*(n_beams-1)*3*2+j*6+1]=v1;
        T[thid*(n_beams-1)*3*2+j*6+2]=v2;
        v2=v1;
        v1=v2-1;
        T[thid*(n_beams-1)*3*2+j*6+3]=v0;
        T[thid*(n_beams-1)*3*2+j*6+4]=v1;
        T[thid*(n_beams-1)*3*2+j*6+5]=v2;
    }
    //__syncthreads(); Not necessary
    //En base a la malla referencial hallamos las demás superficies
    double xp,yp,zp;
    unsigned int count=0,offset,temp_vex,n_triangles_perThreadandDonut=2*(n_beams-1);
    for (unsigned int i = 1; i < n_donuts; i++){
        //Analizamos cada vertice del tríangulo
        for (unsigned int j = 0; j < n_triangles_perThreadandDonut; j++){
             //Analizamos el punto del vertice v0
            offset=(thid*n_triangles_perThreadandDonut+j)*3;
            temp_vex=(T[offset]+i*n_points_perDonut);
            xp=Point_Cloud[temp_vex*3+0];
            yp=Point_Cloud[temp_vex*3+1];
            zp=Point_Cloud[temp_vex*3+2];
            if ((xp!=0)||(yp!=0)||(zp!=0)){
                //analizamos el punto del vertice v1
                temp_vex=(T[offset+1]+i*n_points_perDonut);
                xp=Point_Cloud[temp_vex*3+0];
                yp=Point_Cloud[temp_vex*3+1];
                zp=Point_Cloud[temp_vex*3+2];
                if ((xp!=0)||(yp!=0)||(zp!=0)){
                    //analizamos el punto del vertice v2
                    temp_vex=(T[offset+2]+i*n_points_perDonut);
                    xp=Point_Cloud[temp_vex*3+0];
                    yp=Point_Cloud[temp_vex*3+1];
                    zp=Point_Cloud[temp_vex*3+2];
                    if ((xp!=0)||(yp!=0)||(zp!=0)){
                        //Si todo lo anterior se cumple, guardamos el triángulo
                        T_temp[(n_donuts-1)*n_triangles_perThreadandDonut*3*thid+count*3+2]=temp_vex;
                        T_temp[(n_donuts-1)*n_triangles_perThreadandDonut*3*thid+count*3+1]=(T[offset+1]+i*n_points_perDonut);
                        T_temp[(n_donuts-1)*n_triangles_perThreadandDonut*3*thid+count*3]=(T[offset]+i*n_points_perDonut);
                        count++;
                    }
                }
            }
        }
    }
    count_array[thid]=count;
    __syncthreads();
}
#define ngpu 1024
#define NUM_BANKS 32
#define LOG_NUM_BANKS 5
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS)
__global__ void eScanGPU(unsigned int *g_odata, unsigned int *g_idata)
{
	__shared__ unsigned int temp[2*ngpu];// allocated on invocation
	int thid = threadIdx.x;
	int offset = 1,n=ngpu;
	
	int ai = thid;
	int bi = thid + (n/2);
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
	int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
	temp[ai + bankOffsetA] = g_idata[ai];
	temp[bi + bankOffsetB] = g_idata[bi];

//	temp[2*thid] = g_idata[2*thid]; // load input into shared memory
//	temp[2*thid+1] = g_idata[2*thid+1];
	
	for (int d = n>>1; d > 0; d >>= 1) // build sum in place up the tree
	{
		__syncthreads();
		if (thid < d)
		{
			int ai = offset*(2*thid+1)-1;
			int bi = offset*(2*thid+2)-1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);
//			int ai = offset*(2*thid+1)-1;
//			int bi = offset*(2*thid+2)-1;
			temp[bi] += temp[ai];
		}
		offset *= 2;
	}
	if (thid==0) { temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0; }
	//	if (thid == 0) { temp[n - 1] = 0; } // clear the last element
	for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (thid < d)
		{
			int ai = offset*(2*thid+1)-1;
			int bi = offset*(2*thid+2)-1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);
//			int ai = offset*(2*thid+1)-1;
//			int bi = offset*(2*thid+2)-1;

			unsigned int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();
	g_odata[ai] = temp[ai + bankOffsetA];
	g_odata[bi] = temp[bi + bankOffsetB];
//	g_odata[2*thid] = temp[2*thid]; // write results to device memory
//	g_odata[2*thid+1] = temp[2*thid+1];
}

__global__ void ODF_part2(unsigned int* T,unsigned int* T_temp,unsigned int* count_array,unsigned int* index_offset){
    int thid = threadIdx.x + blockIdx.x * blockDim.x;   
    unsigned int count,offset,n_triangles_perDonut=n_AZBLK*2*(n_beams-1),n_triangles_perThreadandDonut=2*(n_beams-1);
    count=count_array[thid];
    //Ha este punto, cada hilo contiene una cierta cantidad de n triangulos que han de ser colocadas en el array original
    if(thid==0)
    offset=0;
    else
    offset=index_offset[thid]*3;
    //copy triangles 
    for(int z=0;z<count*3;z++){
        T[n_triangles_perDonut*3+offset+z]=T_temp[(n_donuts-1)*n_triangles_perThreadandDonut*3*thid+z];
    }
    __syncthreads();
}

/**
 * \brief Generate surface.  
 */
void Generate_surfaceGPU(double* Point_Cloud,unsigned int* T,unsigned int *pointer_n_triangles){
    /////////////////////////////////////////////////////////////////////////////////////////
    ///////////////////////////////////      CUDA        ////////////////////////////////////
    /////////////////////////////////////////////////////////////////////////////////////////
    hipError_t cudaerr;
    double* Sphere_Cloud_dev;
    hipMalloc((void**)(&Sphere_Cloud_dev), sizeof(double) * n_total_points * 3);
    //1st step. GenerateSphere
    GenerateSphereCUDA<<<4, 256 >>> (Sphere_Cloud_dev);
    //2nd step. Overlap removing
    SupressOverlapCUDA<<<numBlocks, threadsPerBlock >>> (Sphere_Cloud_dev);
    double* Sphere_Cloud;
    Sphere_Cloud = (double*)malloc(n_total_points * 3 *sizeof(double));
    cudaerr=hipMemcpy(Sphere_Cloud, Sphere_Cloud_dev, sizeof(double) *n_total_points * 3, hipMemcpyDeviceToHost);
    if (cudaerr != 0)	printf("ERROR copying to SphereCloud. CudaMalloc value=%i\n\r",cudaerr);
    unsigned int *OneDonutMesh_dev,*T_temp_dev,*count_array_dev,n_triangles_perDonut=n_AZBLK*2*(n_beams-1);
    hipMalloc((void**)(&OneDonutMesh_dev), sizeof(unsigned int) * OneDonutFill_triangles * 3);
    hipMalloc((void**)(&T_temp_dev), sizeof(unsigned int) * n_triangles_perDonut* (n_donuts-1) * 3);
    hipMalloc((void**)(&count_array_dev), sizeof(unsigned int) * 1024);
    //3rd step. First part of the ODF
    ODF_part1<<<numBlocks, threadsPerBlock >>> (Sphere_Cloud_dev,OneDonutMesh_dev,T_temp_dev,count_array_dev);
    unsigned int *index_offset_array_dev;
    hipMalloc((void**)(&index_offset_array_dev), sizeof(unsigned int) * 1024);
    //4th step. eScan GPU
    eScanGPU<<<1, 512 >>> (index_offset_array_dev,count_array_dev);//try with CPU. Try with masking. Chapter: toolkit pin-memory. Optimized method. Chapter: asynch transfer
    //5th step. last part ODF
    ODF_part2<<<numBlocks, threadsPerBlock >>> (OneDonutMesh_dev,T_temp_dev,count_array_dev,index_offset_array_dev);
    unsigned int *T_temp;
    T_temp=(unsigned int*)malloc(n_total_triangles * 3 *sizeof(unsigned int));
    cudaerr=hipMemcpy(T_temp,OneDonutMesh_dev, sizeof(unsigned int) *OneDonutFill_triangles * 3, hipMemcpyDeviceToHost);
    if (cudaerr != 0)	printf("ERROR copying to T_temp. CudaMalloc value=%i\n\r",cudaerr);
	/////////////////////////////////////////////////////////////////////////////////////////
    ///////////////////////////////////      CPU        ////////////////////////////////////
    /////////////////////////////////////////////////////////////////////////////////////////
    //6th step. last Fill
    TwoandTri_Donut_Fill(Sphere_Cloud,&T_temp[OneDonutFill_triangles*3],&T_temp[(OneDonutFill_triangles+TwoDonutFill_triangles)*3],&T_temp[(OneDonutFill_triangles+TwoDonutFill_triangles+TriDonutFill_triangles)*3]);
    hipDeviceSynchronize();
    hipFree(Sphere_Cloud_dev);
    hipFree(OneDonutMesh_dev);
    hipFree(T_temp_dev);
    hipFree(count_array_dev);
    hipFree(index_offset_array_dev);
   
    unsigned int temp_vex,n_triangles=0;
    double xp,yp,zp;
    for (unsigned int i = 0; i < n_total_triangles; i++){
        //Analizamos el punto del vertice v0
        temp_vex=T_temp[i*3];
        xp=Point_Cloud[temp_vex*3+0];
        yp=Point_Cloud[temp_vex*3+1];
        zp=Point_Cloud[temp_vex*3+2];
        if ((xp!=0)&&(yp!=0)&&(zp!=0)){
            //analizamos el punto del vertice v1
            temp_vex=T_temp[i*3+1];
            xp=Point_Cloud[temp_vex*3+0];
            yp=Point_Cloud[temp_vex*3+1];
            zp=Point_Cloud[temp_vex*3+2];
            if ((xp!=0)&&(yp!=0)&&(zp!=0)){
                //analizamos el punto del vertice v2
                temp_vex=T_temp[i*3+2];
                xp=Point_Cloud[temp_vex*3+0];
                yp=Point_Cloud[temp_vex*3+1];
                zp=Point_Cloud[temp_vex*3+2];
                if ((xp!=0)&&(yp!=0)&&(zp!=0)){
                    //Si todo lo anterior se cumple, guardamos el triángulo
                    T[n_triangles*3+2]=temp_vex;
                    T[n_triangles*3+1]=T_temp[i*3+1];
                    T[n_triangles*3]=T_temp[i*3];
                    n_triangles++;
                }
            }
        }
    }
    pointer_n_triangles[0]=n_triangles; 
    free(Sphere_Cloud);
    free(T_temp);
}

/******************************************************************/
/*************************       MAIN     *************************/
/******************************************************************/
#define TESTING 0
int main()
{
#if TESTING == 0
    #define iter 500.0
    /*Allocate memory*/
    double* Point_Cloud;
    Point_Cloud = (double*)malloc(n_total_points * 3 *sizeof(double));
    //Leemos del csv los datos reales
    FILE* archivo;
    archivo = fopen("../files/MinaData.csv", "r");
    char buffer[200];
    char* token;
    //Saltamos la primera línea
    fgets(buffer,sizeof(buffer),archivo);
    for (unsigned int i = 0; i < n_total_points; i++){
        fgets(buffer,sizeof(buffer),archivo);
        token = strtok(buffer,",");
        Point_Cloud[i*3+0]=atof(token);
        token = strtok(NULL,",");
        Point_Cloud[i*3+1]=atof(token);
        token = strtok(NULL,",\n");
        Point_Cloud[i*3+2]=atof(token);
    }    
    fclose(archivo);    
    
    /*****************************************************/
    /********************       CPU     ******************/
    /*****************************************************/
    unsigned int *T_cpu,n_triangles_real_data_cpu;
    T_cpu=(unsigned int*)malloc(n_total_triangles * 3 *sizeof(unsigned int));
    clock_t startCPU;
	clock_t finishCPU;
    printf ("/********************solo CPU*********************/:\n");
	startCPU = clock();
    for (int i=0;i<iter;i++){
        Generate_surface(Point_Cloud,T_cpu,&n_triangles_real_data_cpu);
    }
	finishCPU = clock();
	printf("numero de triangulos: %d\n",n_triangles_real_data_cpu);
    printf("CPU: %fms\n", ((double)(finishCPU - startCPU))*1000/(double)CLOCKS_PER_SEC/iter);
	
    /*****************************************************/
    /********************       GPU     ******************/
    /*****************************************************/
    unsigned int *T_gpu,n_triangles_real_data_gpu;
    T_gpu=(unsigned int*)malloc(n_total_triangles * 3 *sizeof(unsigned int));
    clock_t startGPU;
	clock_t finishGPU;
    printf ("/********************CPU y GPU*********************/:\n");
	startGPU = clock();
    for (int i=0;i<iter;i++){
        Generate_surfaceGPU(Point_Cloud,T_gpu,&n_triangles_real_data_gpu);
    }
	finishGPU = clock();
	printf("numero de triangulos: %d\n",n_triangles_real_data_gpu);
    printf("GPU: %fms\n", ((double)(finishGPU - startGPU))*1000/(double)CLOCKS_PER_SEC/iter);
	//creamos archivo para ver results
    archivo = fopen("../files/CUDAMesh.csv", "w+");
    fprintf(archivo, "V1, V2, V3\n");
    for (unsigned int i=0; i < n_triangles_real_data_gpu; i++) {
        fprintf(archivo,"%d, %d, %d\n", T_gpu[i*3+0], T_gpu[i * 3 + 1], T_gpu[i * 3 + 2]);
    }
    fclose(archivo);
    //------------------------------------------
	//----------Generate the DXF file-----------
	//------------------------------------------
	//Open the DXF file
    archivo = fopen("../files/CUDAMinaSurface.dxf", "w");
    //assert(archivo);
    //header
    fprintf(archivo, "0\nSECTION\n2\nENTITIES\n0\n");
	float x0,y0,z0,x1,y1,z1,x2,y2,z2;
	for (int i = 0; i < n_triangles_real_data_gpu; i++)
    {
        // get the coordinates of each point from the triangle
        x0 = Point_Cloud[T_gpu[i* 3+0]*3 + 0];
        y0 = Point_Cloud[T_gpu[i* 3+0]*3 + 1];
        z0 = Point_Cloud[T_gpu[i* 3+0]*3 + 2];
        
        x1 = Point_Cloud[T_gpu[i* 3+1]*3 + 0];
        y1 = Point_Cloud[T_gpu[i* 3+1]*3 + 1];
        z1 = Point_Cloud[T_gpu[i* 3+1]*3 + 2];
        
        x2 = Point_Cloud[T_gpu[i* 3+2]*3 + 0];
        y2 = Point_Cloud[T_gpu[i* 3+2]*3 + 1];
        z2 = Point_Cloud[T_gpu[i* 3+2]*3 + 2];
        //create new 3DFACE element
        fprintf(archivo, "3DFACE\n8\n1\n");
        fprintf(archivo, " 62\n %d\n", 142);//corresponding color of the autocad pallete
        fprintf(archivo, "10\n %.4f\n 20\n %.4f\n 30\n %.4f\n", x0, y0, z0);
        fprintf(archivo, "11\n %.4f\n 21\n %.4f\n 31\n %.4f\n", x1, y1, z1);
        fprintf(archivo, "12\n %.4f\n 22\n %.4f\n 32\n %.4f\n", x2, y2, z2);
        fprintf(archivo, "13\n %.4f\n 23\n %.4f\n 33\n %.4f\n", x2, y2, z2);
        fprintf(archivo, "0\n");
    }
    fprintf(archivo, "ENDSEC\n 0\nEOF\n");
    fclose(archivo);

    return 0;
#else
    #define iter 100.0
    /*************************************************************************************/
    /***********************************    CPU     **************************************/
    /*************************************************************************************/
    double* Point_Cloud;
    Point_Cloud = (double*)malloc(n_total_points * 3 *sizeof(double));
    unsigned int *T;
    T=(unsigned int*)malloc(n_total_triangles * 3 *sizeof(unsigned int));

    clock_t start,stop;
    double timeCPU=0;
    printf ("/********************solo CPU*********************/:\n");
    start=clock();
    for (int z=0;z<iter;z++)
    Generate_sphere(Point_Cloud);
    stop=clock();
    timeCPU+=((double)(stop - start))*1000.0/(double)CLOCKS_PER_SEC/iter;
    printf("GS time: %fms\n", ((double)(stop - start))*1000.0/(double)CLOCKS_PER_SEC/iter);
    
    start=clock();
    for (int z=0;z<iter;z++)
    Supress_redundant_data(Point_Cloud);
    stop=clock();
    timeCPU+=((double)(stop - start))*1000.0/(double)CLOCKS_PER_SEC/iter;
    printf("OR time: %fms\n", ((double)(stop - start))*1000.0/(double)CLOCKS_PER_SEC/iter);
    
    start=clock();
    for (int z=0;z<iter;z++)
    One_Donut_Fill(Point_Cloud,T);
    stop=clock();
    timeCPU+=((double)(stop - start))*1000.0/(double)CLOCKS_PER_SEC/iter;
    printf("ODF time: %fms\n", ((double)(stop - start))*1000.0/(double)CLOCKS_PER_SEC/iter);
    
    start=clock();
    for (int z=0;z<iter;z++)
    TwoandTri_Donut_Fill(Point_Cloud,&T[OneDonutFill_triangles*3],&T[(OneDonutFill_triangles+TwoDonutFill_triangles)*3],&T[(OneDonutFill_triangles+TwoDonutFill_triangles+TriDonutFill_triangles)*3]);    
    stop=clock();
    timeCPU+=((double)(stop - start))*1000.0/(double)CLOCKS_PER_SEC/iter;
    printf("LastFill time: %fms\n", ((double)(stop - start))*1000.0/(double)CLOCKS_PER_SEC/iter);
    printf("total time: %fms\n",timeCPU);
        
    free(Point_Cloud);
    free(T);
    /*************************************************************************************/
    /***********************************    GPU     **************************************/
    /*************************************************************************************/    
    double* Point_Cloud_dev,*Point_Cloud_gpu;
    hipMalloc((void**)(&Point_Cloud_dev), sizeof(double) * n_total_points * 3);
    Point_Cloud_gpu = (double*)malloc(n_total_points * 3 *sizeof(double));

    hipError_t cudaerr;
    hipEvent_t start_gpu, stop_gpu;
    float timeGPU,totalGPU=0;
    hipEventCreate(&start_gpu);
    hipEventCreate(&stop_gpu);

    printf ("/*********************CPU y GPU********************/\n");
    hipEventRecord(start_gpu);
    for (int z = 0; z < iter; z++)
    GenerateSphereCUDA<<<4, 256 >>> (Point_Cloud_dev);
    hipEventRecord(stop_gpu);
    hipEventSynchronize(stop_gpu);
	hipEventElapsedTime(&timeGPU, start_gpu, stop_gpu);
    printf("GS time:  %fms\n\r", timeGPU / iter);
    totalGPU+=timeGPU/iter;
    hipDeviceSynchronize();
    
    hipEventRecord(start_gpu);
    for (int z = 0; z < iter; z++)
    SupressOverlapCUDA<<<numBlocks, threadsPerBlock  >>> (Point_Cloud_dev);
    cudaerr=hipMemcpy(Point_Cloud_gpu, Point_Cloud_dev, sizeof(double) *n_total_points * 3, hipMemcpyDeviceToHost);
    if (cudaerr != 0)	printf("ERROR copying to Point_Cloud_gpu. CudaMalloc value=%i\n\r",cudaerr);
    hipEventRecord(stop_gpu);
    hipEventSynchronize(stop_gpu);
	hipEventElapsedTime(&timeGPU, start_gpu, stop_gpu);
    printf("OR time:  %fms\n\r", timeGPU / iter);
    totalGPU+=timeGPU/iter;

    unsigned int *T_gpu,*OneMesh_dev,*OneMesh_temp_dev,n_triangles_perDonut=n_AZBLK*2*(n_beams-1);
    unsigned int *count_array_dev,*index_offset_array_dev;
    
    hipMalloc((void**)(&OneMesh_dev), sizeof(unsigned int) * n_total_triangles * 3);
    hipMalloc((void**)(&OneMesh_temp_dev), sizeof(unsigned int) * n_triangles_perDonut* (n_donuts-1) * 3);
    hipMalloc((void**)(&count_array_dev), sizeof(unsigned int) * 1024);
    hipMalloc((void**)(&index_offset_array_dev), sizeof(unsigned int) * 1024);
    T_gpu=(unsigned int*)malloc(n_total_triangles * 3 *sizeof(unsigned int));

    //3rd step. First part of the ODF
    hipDeviceSynchronize();
    hipEventRecord(start_gpu);
    for (int z = 0; z < iter; z++){
        ODF_part1<<<numBlocks, threadsPerBlock >>> (Point_Cloud_dev,OneMesh_dev,OneMesh_temp_dev,count_array_dev);
        hipDeviceSynchronize();
        //4th step. eScan GPU
        eScanGPU<<<1, 1024/2 >>> (index_offset_array_dev,count_array_dev);
        hipDeviceSynchronize();
        //5th step. last part ODF
        ODF_part2<<<numBlocks, threadsPerBlock >>> (OneMesh_dev,OneMesh_temp_dev,count_array_dev,index_offset_array_dev);
        hipDeviceSynchronize();
        cudaerr=hipMemcpy(T_gpu,OneMesh_dev, sizeof(unsigned int) *OneDonutFill_triangles * 3, hipMemcpyDeviceToHost);
    
    }
    hipEventRecord(stop_gpu);
    hipEventSynchronize(stop_gpu);
	hipEventElapsedTime(&timeGPU, start_gpu, stop_gpu);
    if (cudaerr != 0)	printf("ERROR copying to T_gpu. CudaMalloc value=%i\n\r",cudaerr);
    printf("ODF time:  %fms\n\r", timeGPU / iter);
    totalGPU+=timeGPU/iter;
  
    start=clock();
    for (int z=0;z<iter;z++)
    TwoandTri_Donut_Fill(Point_Cloud_gpu,&T_gpu[OneDonutFill_triangles*3],&T_gpu[(OneDonutFill_triangles+TwoDonutFill_triangles)*3],&T_gpu[(OneDonutFill_triangles+TwoDonutFill_triangles+TriDonutFill_triangles)*3]);    
    stop=clock();
    totalGPU+=((double)(stop - start))*1000.0/(double)CLOCKS_PER_SEC/iter;
    printf("LastFill time: %fms\n", ((double)(stop - start))*1000.0/(double)CLOCKS_PER_SEC/iter);
    
    printf("total time: %fms\n",totalGPU);

    
    FILE* archivo;
    archivo = fopen("../testfiles/CUDASphere_cloud.csv", "w+");
    fprintf(archivo, "X, Y, Z\n");
    for (unsigned int i=0; i < n_total_points; i++) {
        fprintf(archivo,"%.4f, %.4f, %.4f\n", Point_Cloud_gpu[i*3+0], Point_Cloud_gpu[i * 3 + 1], Point_Cloud_gpu[i * 3 + 2]);
    }
    fclose(archivo);
    archivo = fopen("../testfiles/CUDAOneMesh.csv", "w+");
    fprintf(archivo, "V1, V2, V3\n");
    for (unsigned int i=0; i < n_total_triangles; i++) {
        fprintf(archivo,"%d, %d, %d\n", T_gpu[i*3+0], T_gpu[i * 3 + 1], T_gpu[i * 3 + 2]);
    }
    fclose(archivo);
    printf("fin\n");
    return;
  



    //Finalmente, liberamos el resto de memoria
    hipFree(Point_Cloud_dev);
    hipFree(OneMesh_dev);
    hipFree(OneMesh_temp_dev);
    hipFree(count_array_dev);
    hipFree(index_offset_array_dev);
    free(Point_Cloud_gpu);
    free(T_gpu); 
    return 0;
#endif
}
