#include <stdio.h>
#include <string.h>
#include <stdint.h>
#include <stdlib.h>
#include "math.h"
#include "time.h"
#ifndef MPI//definimos el valor de PI
#define MPI 3.14159265358979323846
#define MPI_2 1.57079632679489661923
#endif
#define D180_MPI 0.017453293 //: Degrees180/pi
/*----------------------------------------------------------------------------*/
/**
 * Definimos los parámetros del LIDAR
 * Se puede usar typedef para juntar los parámetros del LIDAR, o mejor no.
 */
/*Cantidad de rayos por azimuth*/
#define n_beams 16
/*Cantidad de azimuths por Donut*/
#define n_AZBLK 1024
/*Los ángulos por defecto de fabrica del LIDAR*/
double beam_altitude_angles[n_beams]= {15.379*D180_MPI,13.236*D180_MPI,11.128*D180_MPI,9.03*D180_MPI,6.941*D180_MPI,4.878*D180_MPI,2.788*D180_MPI,0.705*D180_MPI,-1.454*D180_MPI,-3.448*D180_MPI,-5.518*D180_MPI,-7.601*D180_MPI,-9.697*D180_MPI,-11.789*D180_MPI,-13.914*D180_MPI,-16.062*D180_MPI};
double beam_azimuth_angles[n_beams] = { -1.24*D180_MPI, -1.2145*D180_MPI, -1.1889*D180_MPI, -1.1634*D180_MPI, -1.1379*D180_MPI, -1.1123*D180_MPI, -1.0868*D180_MPI, -1.0613*D180_MPI, -1.0357*D180_MPI, -1.0102*D180_MPI, -0.98467*D180_MPI, -0.95913*D180_MPI, -0.9336*D180_MPI, -0.90807*D180_MPI, -0.88253*D180_MPI, -0.857*D180_MPI };
/*Cantidad puntos de la nube*/
#define n_points_perDonut (unsigned int)(n_AZBLK*n_beams)
/*Angulo entre azimuths*/
#define angle_between_azimuths (-2*MPI/n_AZBLK)
/*Ángulo de rotacion del motor*/
#define rot_angle (-33.53706667 * MPI / 180)
/*Creamos la matrix de rotación del eje del motor (eje Z)*/
double rot_motor_matrix[9]={ cos(rot_angle),-sin(rot_angle),0,sin(rot_angle),cos(rot_angle) ,0,0,0,1 };
/*Cantidad de Donuts en función del ángulo de rotación*/
#define n_donuts 6//(unsigned int) ceil(-MPI/rot_angle)
#define n_total_points (unsigned int)(n_donuts*n_points_perDonut)
/*Cantidad de triángulos*/
#define OneDonutFill_triangles          108226
#define TwoDonutFill_triangles          6883
#define TriDonutFill_triangles          360
#define MidDonutFill_triangles          2793
#define n_total_triangles (OneDonutFill_triangles+TwoDonutFill_triangles+TriDonutFill_triangles+MidDonutFill_triangles)

//CUDA libraries
#define threadsPerBlock 1024
#define numBlocks (n_AZBLK/1024)
#include "hip/hip_runtime.h"

#ifndef __CUDACC__  
	#define __CUDACC__
	#include <device_functions.h>
#endif

/*----------------------------------------------------------------------------*/
/**
 * Funciones de conversión
 */
void rad2deg(double *value) {
    *value = (*value) * 180 / MPI;
}
void deg2rad(double *value) {
    value[0] = value[0] * MPI / 180;
}
void mult_matrix(double* A,unsigned int m,unsigned int n,double*B,unsigned int l,double*C) {
    for (unsigned int i = 0; i < l;i++) {
        for (unsigned int j = 0; j < m; j++) {
            C[j*l+i]=0;
            for (unsigned int k = 0; k < n; k++) {
                C[j * l +i] += A[j*n+k]*B[k*l + i];
            }
        }
    }
}
/*----------------------------------------------------------------------------*/
/** 
 * Funciones de matrices de rotación
 */
void rot_x_axis(double* XYZ_points,double angle ){
    double temp[3], rot_matrix[9] = { 1,0,0,0,cos(angle),-sin(angle),0,sin(angle),cos(angle) };
    memcpy(temp, XYZ_points, 3*sizeof(double));
    mult_matrix(rot_matrix, 3, 3, temp, 1, XYZ_points);
}
void rot_y_axis(double* XYZ_points, double angle) {
    double temp[3], rot_matrix[9] = { cos(angle),0,sin(angle),0,1,0,-sin(angle),0,cos(angle) };
    memcpy(temp, XYZ_points, 3 * sizeof(double));
    mult_matrix(rot_matrix, 3, 3, temp, 1, XYZ_points);
}
void rot_z_axis(double* XYZ_points, double angle) {
    double temp[3], rot_matrix[9] = { cos(angle),-sin(angle),0,sin(angle),cos(angle) ,0,0,0,1};
    memcpy(temp, XYZ_points, 3 * sizeof(double));
    mult_matrix(rot_matrix, 3, 3, temp, 1, XYZ_points);
}
/*----------------------------------------------------------------------------*/
/**
 * \brief Generate synthetic sphere. La generación se divide en 3 estapas. La 
 * 1ra etapa consta en la generación de un azimut referencial; para ello, se 
 * define que el azimuth referencial será aquel que se genera en la vertical
 * inferior, luego, ubicamos cada punto en el plano XZ y luego le realizamos la
 * rotación en el eje X debido al desfase de rayos en cada azimut. En la 2da 
 * etapa se realiza el barrido del azimut referencial para obtener la Donut 
 * refernecial, y en la 3era etapa se realiza la rotación a la Donut referencial
 * para obtener la esfera modelada.
 * 
 * \param Point_Cloud es el puntero que tendrá los puntos de la nube de puntos
 * de la esfera sintética modelada
 * 
 * \return None
 */
#define Radius_sphere 1.0
void Generate_sphere(double* Point_Cloud) {
    /**
    * Nuestro sistema de referencia será : Eje Z será el eje de giro del motor.
    *                                      Eje X será el eje de la Donut referencial
    */
    double R= Radius_sphere;
    /*Generamos el azimuth refencial y los azimuts de cada sector*/
    for (int i = 0; i < n_beams; i++) {
        //ubicamos el punto del azimut referencial en el plano XZ
        Point_Cloud[3 * i + 0] = R* cos(beam_altitude_angles[i] - MPI_2); //x
        Point_Cloud[3 * i + 1] = 0;                                         //y
        Point_Cloud[3 * i + 2] = R * sin(beam_altitude_angles[i] - MPI_2);//z
        //Realizamos la rotacion del punto con respecto al eje x debido al desfase
        rot_x_axis(&Point_Cloud[3*i],beam_azimuth_angles[i]);
        /*Creamos los azimuts que inician en cada sector*/
        /*mirror points from quarter Donut*/
        Point_Cloud[(i + n_AZBLK / 4 * n_beams)*3+0] = Point_Cloud[3 * i + 0];
        Point_Cloud[(i + n_AZBLK / 4 * n_beams)*3+1] = Point_Cloud[3 * i + 2];
        Point_Cloud[(i + n_AZBLK / 4 * n_beams)*3+2] = -Point_Cloud[3 * i + 1];
        /*mirror points from midle Donut*/
        Point_Cloud[(i + n_AZBLK / 2 * n_beams)*3+0] = Point_Cloud[3 * i + 0];
        Point_Cloud[(i + n_AZBLK / 2 * n_beams)*3+1] = -Point_Cloud[3 * i + 1];
        Point_Cloud[(i + n_AZBLK / 2 * n_beams)*3+2] = -Point_Cloud[3 * i + 2];
        /*mirror points from 3 quater Donut*/
        Point_Cloud[(i + n_AZBLK * 3 / 4 * n_beams)*3 +0] = Point_Cloud[3 * i + 0];
        Point_Cloud[(i + n_AZBLK * 3 / 4 * n_beams)*3 +1] = -Point_Cloud[3 * i + 2];
        Point_Cloud[(i + n_AZBLK * 3 / 4 * n_beams)*3 +2] = Point_Cloud[3 * i + 1];

    }
    /*Definimos la matrix de rotación para los azimuth*/
    double rot_matrix[9] = { 1,0,0,0,cos(angle_between_azimuths),-sin(angle_between_azimuths),0,sin(angle_between_azimuths),cos(angle_between_azimuths) };
    /*Procedemos a realizar el barrido de cada sector para obtener la donnut referencial*/
    double XYZ[3],temp[9];
    for (int i = 1; i < n_AZBLK / 4; i++) {
        for (int j = 0; j < n_beams;j++) {
            //Calculate previous point
            XYZ[0] = Point_Cloud[((i - 1) * n_beams + j)*3 + 0];
            XYZ[1] = Point_Cloud[((i - 1) * n_beams + j)*3 + 1];
            XYZ[2] = Point_Cloud[((i - 1) * n_beams + j)*3 + 2];
            //rotate that point
            mult_matrix (rot_matrix,3,3,XYZ,1,temp);
            //Set the new azimuth
            Point_Cloud[(i * n_beams + j)*3 + 0] = temp[0];
            Point_Cloud[(i * n_beams + j)*3 + 1] = temp[1];
            Point_Cloud[(i * n_beams + j)*3 + 2] = temp[2];
            //mirror from quarter Donunt
            Point_Cloud[((i + n_AZBLK / 4) * n_beams + j)*3 + 0] = temp[0];
            Point_Cloud[((i + n_AZBLK / 4) * n_beams + j)*3 + 1] = temp[2];
            Point_Cloud[((i + n_AZBLK / 4) * n_beams + j)*3 + 2] = -temp[1];
            //mirror points from midle Donut
            Point_Cloud[((i + n_AZBLK / 2) * n_beams + j)*3+0] = temp[0];
            Point_Cloud[((i + n_AZBLK / 2) * n_beams + j)*3+1] = -temp[1];
            Point_Cloud[((i + n_AZBLK / 2) * n_beams + j)*3+2] = -temp[2];
            //mirror points from midle Donut
            Point_Cloud[((i + n_AZBLK * 3 / 4) * n_beams + j)*3 + 0] = temp[0];
            Point_Cloud[((i + n_AZBLK * 3 / 4) * n_beams + j)*3 + 1] = -temp[2];
            Point_Cloud[((i + n_AZBLK * 3 / 4) * n_beams + j)*3 + 2] = temp[1];
        }
    }
    //Rotamos la Donut referencial
    for (unsigned int i = 1; i < n_donuts; i++) {
        //multiplicamos a todos los n_point_perdonut anteriores con la matriz de rotación
        for (unsigned int j = 0; j < n_points_perDonut; j++) {
            mult_matrix(rot_motor_matrix, 3, 3, &Point_Cloud[((i - 1) * n_points_perDonut + j) * 3], 1, temp);
            Point_Cloud[(i * n_points_perDonut + j) * 3 + 0] = temp[0];
            Point_Cloud[(i * n_points_perDonut + j) * 3 + 1] = temp[1];
            Point_Cloud[(i * n_points_perDonut + j) * 3 + 2] = temp[2];
;        }
    }
}
/*----------------------------------------------------------------------------*/
/**
 * \brief Ecuación de la recta. Dado los parametros de una recta, se devuelve el
 * valor del punto y que corresponde al punto x
 * 
 * \param m.Es la pendiente de la recta
 * \param x.Es el punto x al cual se evalua para obtener el punto y correspondiente
 * \param xb.Punto xo donde pasa la recta
 * \param yb.Punto yo donde pasa la recta
  * 
 * \return y. resultado de realizar la ecuación
 */
double eq_line(double m,double x,double xb,double yb) {
    double y= m*(x-xb)+yb;
    return y;
}
/*----------------------------------------------------------------------------*/
/**
 * \brief Supress redundant data. En esta función se realiza la supresión de los 
 * puntos. Primero se debe definir los límites de la donut referencial y con 
 * respecto a las siguientes donuts se analiza el límite de la donut anterior. 
 * Definidos los límites, se eliminan aquello puntos que se encuentren en zonas 
 * no permitidas 
 * 
 * \param Point_Cloud es el puntero que tendrá los puntos de la nube de puntos ha 
 * ser reducida 
 * 
 * \return None
 */
void Supress_redundant_data(double* Point_Cloud){
    //Declare tomporary variables
    double x,y,y_temp,x1,y1,m;//xn,yn
    //Set vertical limits
    double X_L1=Point_Cloud[0],X_Ln=Point_Cloud[(n_beams-1)*3+0];
    //Array which contains lineś parameters
    double L[(n_donuts-2)*5];
    //Declare parameters from Donut_2 to Donut_n-1. Ya que para la supresión solo
    //se necesita los parametros de la Donut anterior y no necesitamos el de la
    //última Donut
    for (int i = 1; i < n_donuts-1; i++)
    {
        //Hallamos dos puntos de la recta proyectada en el plano XY
        x1=Point_Cloud[i*n_points_perDonut*3+0];
        y1=Point_Cloud[i*n_points_perDonut*3+1];
        x=Point_Cloud[i*n_points_perDonut*3+n_beams*3+0];
        y=Point_Cloud[i*n_points_perDonut*3+n_beams*3+1];
        //Calculamos la pendiente
        m=(y-y1)/(x-x1);
        //Guardamos los valores de un punto de la recta del beam_0
        L[(i-1)*5+0]=x1;
        L[(i-1)*5+1]=y1;
        //Guardamos los valores de un punto de la recta del beam_n
        L[(i-1)*5+2]=Point_Cloud[i*n_points_perDonut*3+n_beams*3-3];//xn
        L[(i-1)*5+3]=Point_Cloud[i*n_points_perDonut*3+n_beams*3-3+1];//yn
        //Guardamos el valor de la pendiente hallada
        L[(i-1)*5+4]=m;
    }
    //Supress redundant data only for Donut 2
    int i=1;
    for (unsigned int j = 0; j < n_points_perDonut; j++){
        //Calculamos si la coordenada del punto x
        x=Point_Cloud[i*n_points_perDonut*3+j*3];
        //Analizamos si se encuentra en la zona de supresión
        if (X_Ln<=x&&x<=X_L1){
            Point_Cloud[i*n_points_perDonut*3+j*3]=0;
            Point_Cloud[i*n_points_perDonut*3+j*3+1]=0;
            Point_Cloud[i*n_points_perDonut*3+j*3+2]=0;
            //points_deleted=points_deleted+1;
        }
    }
    //Supress redundant for the rest of the Donuts
    //Creamos variable booleana para saber la zona del plano en donde se encuentra el punto
    bool left_side;
    for (unsigned int i = 2; i < n_donuts; i++){
        for (unsigned int j = 0; j < n_points_perDonut; j++){
            //Hallamos las coordenadas del punto a analizar
            x=Point_Cloud[i*n_points_perDonut*3+j*3];
            y=Point_Cloud[i*n_points_perDonut*3+j*3+1];
            //Evaluamos si se encuentra en la zona referencial
            if (X_Ln<=x){
                if(x<=X_L1){
                    Point_Cloud[i*n_points_perDonut*3+j*3]=0;
                    Point_Cloud[i*n_points_perDonut*3+j*3+1]=0;
                    Point_Cloud[i*n_points_perDonut*3+j*3+2]=0;
                    continue;
                }else{
                    //Se encuentra del lado derecho
                    left_side=false;
                }
            }else{
                //Se encuentra del lazo izquierdo
                left_side=true;
            }
            //Calculamos el valor de y_temp el cual limitará la zona
            y_temp=eq_line(L[(i-2)*5+4],x,L[(i-2)*5+left_side*2],L[(i-2)*5+left_side*2+1]);
            //Le colocamos un signo negativo, o no, para poder realizar un único condicional para ambos casos
            y_temp=y_temp*(1.0-2*left_side);
            y=y*(1.0-2*left_side);
            //Evaluamos la condición de supresión
            if (y>=y_temp){
                //Eliminamos los puntos
                Point_Cloud[i*n_points_perDonut*3+j*3]=0;
                Point_Cloud[i*n_points_perDonut*3+j*3+1]=0;
                Point_Cloud[i*n_points_perDonut*3+j*3+2]=0;
                //points_deleted=points_deleted+1;
            }
        }   
    }
}
/*----------------------------------------------------------------------------*/
/**
 * \brief One-Donut-Fill. Dado una nube de puntos sin traslape y siguiendo el 
 * patrón de medición definido por el sistema Ouster-motor, se obtiene como 
 * salida la malla triangular de las superficies que pertenecen a una única Donut.
 * Primero se realiza la triangulación de la Donut referencial, luego en base a 
 * esta triangulación se obtiene la de las demás Donuts, pero verificando si los
 * vértices son distintos de cero
 * 
 * \param Point_Cloud es el puntero que tendrá los puntos de la nube de puntos  
 * 
 * \param T es el puntero donde se almacenará los vértices de los triángulos
 * 
 * \return None
 */
#define mask (n_points_perDonut-1)
void One_Donut_Fill(double* Point_Cloud,unsigned int* T){
    unsigned int v0,v1,v2;
    //Definimos los vértices
    //Realizamos la malla triangular para la Donut referencial
    for (unsigned int j = 0; j < n_AZBLK; j++){
        for (unsigned int k = 0; k < n_beams-1; k++)
        {   
            v0=j*n_beams+k;
            v2=v0+1;
            v1=(v0+n_beams+1)&mask;
            T[j*(n_beams-1)*3*2+k*6]=v0;
            T[j*(n_beams-1)*3*2+k*6+1]=v1;
            T[j*(n_beams-1)*3*2+k*6+2]=v2;
            v2=v1;
            v1=v2-1;
            T[j*(n_beams-1)*3*2+k*6+3]=v0;
            T[j*(n_beams-1)*3*2+k*6+4]=v1;
            T[j*(n_beams-1)*3*2+k*6+5]=v2;
        }
    }
    //En base a la malla referencial hallamos las demás superficies
    double xp,yp,zp;
    unsigned int temp_vex,count=0,n_triangles_perDonut=n_AZBLK*2*(n_beams-1);
    for (unsigned int i = 1; i < n_donuts; i++){
        //Analizamos cada vertice del tríangulo
        for (unsigned int j = 0; j < n_triangles_perDonut; j++){
            //Analizamos el punto del vertice v0
            temp_vex=(T[j*3]+i*n_points_perDonut);
            xp=Point_Cloud[temp_vex*3+0];
            yp=Point_Cloud[temp_vex*3+1];
            zp=Point_Cloud[temp_vex*3+2];
            if ((xp!=0)||(yp!=0)||(zp!=0)){
                //analizamos el punto del vertice v1
                temp_vex=(T[j*3+1]+i*n_points_perDonut);
                xp=Point_Cloud[temp_vex*3+0];
                yp=Point_Cloud[temp_vex*3+1];
                zp=Point_Cloud[temp_vex*3+2];
                if ((xp!=0)||(yp!=0)||(zp!=0)){
                    //analizamos el punto del vertice v2
                    temp_vex=(T[j*3+2]+i*n_points_perDonut);
                    xp=Point_Cloud[temp_vex*3+0];
                    yp=Point_Cloud[temp_vex*3+1];
                    zp=Point_Cloud[temp_vex*3+2];
                    if ((xp!=0)||(yp!=0)||(zp!=0)){
                        //Si todo lo anterior se cumple, guardamos el triángulo
                        T[n_triangles_perDonut*3+count*3+2]=temp_vex;
                        T[n_triangles_perDonut*3+count*3+1]=(T[j*3+1]+i*n_points_perDonut);
                        T[n_triangles_perDonut*3+count*3]=(T[j*3]+i*n_points_perDonut);
                        count++;
                    }
                }
            }
        }
    }
    //printf("numero de triangulos: %d\n",count+n_triangles_perDonut);    
}
/*----------------------------------------------------------------------------*/
/**
 * \brief get_tripivot. Dado el punto del vértice de la Donut, se hallan los otros
 * vértices del triángulo pívot. 
 * 
 * \param v0_pointer.  puntero del vértice v0
 * 
 * \param v1_pointer. puntero del vértice v1
 * 
 * \param point. Puntero que contiene las coordenadas del vertice v2 del tripivot
 * 
 * \param sector. Indica en que sector estamos de la Donut
 * 
 * \param i. Indica para que Donut se está hallando el tripivot
 * 
 * \param k_beam. Nos dice si nos enfocamos con el beam=0 o el beam=15
 * 
 * \return None
 */
void get_tripivot(unsigned int *vmin_pointer,unsigned int *vmax_pointer,double *point,unsigned int sector,unsigned int i,unsigned int k_beam){
    //Definimos las variables
    unsigned int offset,k_azimuth,vmin,vmax;
    double y_data,z_data,theta,rot_theta,alfa;
    double rot_point[3];
    //Se debe tener en cuenta los rango de acos y asin
    //  acos: [0~pi]
    //  atan: [-pi/2~pi/2]
    rot_point[0]=point[0];
    rot_point[1]=point[1];
    rot_point[2]=point[2];
    //Con el valor de x_data, podemos hallar el ángulo del punto con respecto a 
    //la Donut referencial, teniendo como eje de giro el eje z
    theta=-acos(point[0]/Radius_sphere);
    //Asimismo, debemos hallar el ángulo con respecto al al Donut previa, para ello
    //realizamos un antigiro para que la Donut previa "aparente" ser la referencial y
    //usar las funciones ya establecidas
    rot_z_axis(rot_point, -rot_angle*(i-1));
    rot_theta=-acos(rot_point[0]/Radius_sphere);
    if (theta*(1.0-2.0*((sector>>1)&0x1)) >= rot_theta*(1.0-2*((sector>>1)&0x1)))
    {
        //Los vértices pertenecen a la Donut referencias
        y_data=point[1];
        z_data=point[2];
        offset=0;
    }else{
        //los vértices pertenecerán a la previa Donut
        y_data=rot_point[1];
        z_data=rot_point[2];
        theta=rot_theta;
        offset=n_AZBLK*n_beams*(i-1);
    }
    //Hallamos el ángulo alfa, que es el angulo del azimuth
    alfa=-MPI_2 +atan(z_data/y_data);
    if (y_data>0) alfa=-MPI+alfa;
    //Añadimos -2pi al alfa para no tener probelmas con el bitwise and,
    //esto al final no perjudica ya que se hace el masking de bits
    //solo que tener en cuenta que la división entre -2pi/ang_bet_azit
    //da un total de 1024.
    alfa=alfa-2*MPI;
    //Con lo anterior, nos hemos asegurado que el alfa sea siempre negativo
    //Calculamos el azimuth que corresponde al alfa
    k_azimuth=(unsigned int)((alfa-beam_azimuth_angles[k_beam])/angle_between_azimuths);
    vmin=(k_azimuth)*n_beams+k_beam;
    //Enmascaramos
    vmin=vmin&mask;
    vmax=(vmin+n_beams)&mask;
    //Realizamos offset y sentido
    vmin+=offset;
    vmax+=offset;
    *vmin_pointer=vmin;
    *vmax_pointer=vmax;
                
}
/*----------------------------------------------------------------------------*/
/**
 * \brief side2sideFill. Dado una nube de puntos sin traslape y siguiendo 
 * el patrón de medición definido por el sistema Ouster-motor, se obtiene como 
 * 
 * \param parameter-name description
 * 
 * \return None
 */
void side2sideFill( unsigned int vL0_init,unsigned int vL1_init,
                    unsigned int vL0_fin,unsigned int vL1_fin,
                    int pasoL0,int pasoL1,
                    unsigned int* T,unsigned int* n_triangles_pointer)
{
    unsigned int vmax,vmin,v0,v1,v2,v1_fin,v_temp;//,v2_fin;
    int freepointsL0,freepointsL1,pasov1,pasov2,arista_mismo_vex;
    //Realizamos algunos ajustes para poder obterner el valor magnitud de
    //los puntos libres tanto para la izquierda y derecha
    if (pasoL0<0){
        vmax=vL0_init&mask;
        vmin=vL0_fin&mask;
    }else{
        vmax=vL0_fin&mask;
        vmin=vL0_init&mask;
    }
    freepointsL0=((vmax+(mask-vmin)+1)&mask)/abs(pasoL0);
    if (pasoL1<0){
        vmax=vL1_init&mask;
        vmin=vL1_fin&mask;
    }else{
        vmax=vL1_fin&mask;
        vmin=vL1_init&mask;
    }
    freepointsL1=((vmax+(mask-vmin)+1)&mask)/abs(pasoL1);
    //Con las siguientes formulas podemos hallar con que Donut 
    //estamos trabajando y obtener el offset adecuado
    unsigned int offsetL0=n_points_perDonut*floor(double((vL0_init-1))/n_points_perDonut);
    unsigned int offsetL1=n_points_perDonut*floor(double((vL1_init-1))/n_points_perDonut);
    unsigned int offset;
    //considero que "v2 es de L0" y "v1 es de L1"
    v2=vL0_init;
    v1=vL1_init;
    //v2_fin=vL0_fin;
    v1_fin=vL1_fin;
    pasov2=pasoL0;
    pasov1=pasoL1;
    //la siguiente variable es para el caso que el vertice en comun este en L2
    bool volteamos=false;
    //Analizamos si tenemos distintos puntos
    if (freepointsL1!=freepointsL0){
        //Realizamos un triangulo con mismo vertice
        arista_mismo_vex=freepointsL1-freepointsL0;
        offset=offsetL1;
        if (arista_mismo_vex<0){
            //En caso L1 tenga mas puntos el v_comun estará en L2
            v2=vL1_init;
            v1=vL0_init;
            pasov1=pasoL0;
            offset=offsetL0;
            arista_mismo_vex=arista_mismo_vex*-1;
            volteamos=true;
        }
        for (unsigned int j=0;j<arista_mismo_vex;j++){
            v0=v1;
            //debemos realizar el offset adecuado segun la Donut con la que
            //trabajamos. Primero le sumamos el mask y luego el paso, esto para obtener
            //la concatenación en la misma Donut. Luego le sumamos el offset de la Donut
            //le sumamos la unidad para que luego de hacer el bitmasking obtengamos el mismo
            //numero
            v1=(((v1+mask+1)+pasov1)&mask)+offset;
            T[n_triangles_pointer[0]*3+0]=v0;
            T[n_triangles_pointer[0]*3+1]=v1;
            T[n_triangles_pointer[0]*3+2]=v2;
            n_triangles_pointer[0]++;
        }
        if (volteamos){
            //en caso habiamos volteado, volvemos al caso inicial
            v_temp=v2;
            v2=v1;
            v1=v_temp;
            pasov1=pasoL1;
        }
    }
    //Realizamos el llenado "alineado"
    while (v1!=v1_fin){
        v0=v1;
        //Con esta formula podemos obtener la concatenación de Donuts
        v1=(((v2+mask+1)+pasov2)&mask)+offsetL0;
        T[n_triangles_pointer[0]*3+0]=v0;
        T[n_triangles_pointer[0]*3+1]=v1;
        T[n_triangles_pointer[0]*3+2]=v2;
        n_triangles_pointer[0]++;
        v2=v1;
        //Esta formula permite avanzar por medio del enmascaramiento sin
        //necesidad de usar condicionales (equivalente al operador modulo)
        v1=(((v0+mask+1)+pasov1)&mask)+offsetL1;
        T[n_triangles_pointer[0]*3+0]=v0;
        T[n_triangles_pointer[0]*3+1]=v1;
        T[n_triangles_pointer[0]*3+2]=v2;
        n_triangles_pointer[0]++;
    }
}

/*----------------------------------------------------------------------------*/
/**
 * \brief Two-and-Tri-Donut-Fill. Dado una nube de puntos sin traslape y siguiendo 
 * el patrón de medición definido por el sistema Ouster-motor, se obtiene como 
 * salida la malla triangular de las superficies que pertenecen a dos y tres Donut.
 * Para realizar esto, debemos seguir una serie de pasos:
 * 
 * Primero, se debe hallar los triángulos pivots, pero para ello se deben hallar
 * los triángulos de cada sector. Es por eso que primero establecemos unos valores
 * de índice inicial el cual estarán ubicado en el "centro de la esfera, estos 
 * valores de indice serán aumentados/reducidos para hallar el vértice no nulo que
 * formará parte del triángulo pivot.
 * 
 * Luego,
 * 
 * \param Point_Cloud es el puntero que tendrá los puntos de la nube de puntos  
 * 
 * \param T es el puntero donde se almacenará los vértices de los triángulos
 * 
 * \return None
 */
void TwoandTri_Donut_Fill(double* Point_Cloud,unsigned int* TwoDF,unsigned int* TriDF,unsigned int* MidDF){
    int paso;
    unsigned int n_tripivot,init_index,k_beam;
    unsigned int n_Twotriangles=0,n_Tritriangles=0,n_Midtriangles=0;
    double x_point,y_point,z_point;
    unsigned int Tripivot[16*3];
    //Creamos el arreglo que contendrá a los cuatro triángulos de cada sector que limitan la zona del medio
    unsigned int Tripivot_middle[4*3];
    //Para la última Donut se deben almacenar otros 4 triángulos
    unsigned int Tripivot_middle_particular[4*3];
    //last_tripivots almacena los últimos triángulos pivots de lso 4 sectores
    unsigned int last_Tripivots[4*3]={0,0,0,0,0,0,0,0,0,0,0,0};
    bool vex_found;
    //variables para clasificar la zona de llenado
    char escalera,tipo;
    //Variables de paso
    int pasoL0,pasoL1;
    //Variables para operar con los vértices
    unsigned int v0,v1,v2,v_temp,v_corner,v_corner_fin,v_init,v_fin;
    //Variables para operar con los vertices en el TriDonutFill
    unsigned int v0_mid,v1_mid,v2_mid,T_mid[3],v0_lim,v1_lim,index;//v2_lim,new_v_init;
    int paso_mid,freepoints_mid;
    unsigned int a,b,c;
    //Variables para el middlefill
    unsigned int vrigth_init,vrigth_fin,vleft_init,vleft_fin;
    int pasoRigth,pasoLeft;
    //A partir de la segunda Donut generamos las superficies
    for (int i = 1; i < n_donuts; i++){
        for (unsigned int sector = 0; sector < 4; sector++){
            n_tripivot=0;
            /**************************************************************************************/
            /****************************   Creamos los tripivots   *******************************/
            /**************************************************************************************/
            for (unsigned int j = 0; j < n_beams; j++){
                //Debemos hallar el vertice límite distinto de cero, para ello usamos un 
                //init_index que nos permitirá evaluar cada punto hasta hallar el que es 
                //distinto de cero. Para hacer esto, el init_index debe iniciar en la parte
                //inferior o superior de la esfera, es decir, donde los puntos son igual a 
                //cero. init_index será (n_points-n_beams)+j o (n_points/2-n_beams)+j, luego
                //se le añade el offset correspondiente a cada Donut.
                //Segun el sector, podemos definir si debería iniciar en la parte superior o
                //inferior:
                init_index=(n_AZBLK>>1)*(2-(((sector>>1)&0x1)^(sector&0x1)))-1;
                init_index=init_index*n_beams+j;
                //Definimos si el paso será positivo o negativo, esto dependerá del sector.
                paso=n_beams*(1.0-2*(sector&0x1));
                //Creamos bandera para saber si se halló un vértice para crear el tripivot
                vex_found=false;
                for (unsigned int count = 0; count < (n_AZBLK>>2); count++){
                    //Entramos en un bucle donde evaluamos los puntos de la Donut hasta hallar
                    //el punto que sea distinto de cero
                    x_point=Point_Cloud[i*n_points_perDonut*3+init_index*3+0];  
                    y_point=Point_Cloud[i*n_points_perDonut*3+init_index*3+1];
                    z_point=Point_Cloud[i*n_points_perDonut*3+init_index*3+2];
                    //Evaluamos condición
                    if ((x_point!=0)||(y_point!=0)||(z_point!=0)){
                        //Hallamos vertice distinto de cero
                        vex_found=true;
                        //Agregamos el offset de la Donut correspondiente
                        v2=init_index+i*n_points_perDonut;
                        break;
                    }
                    //Realizamos el paso correspondiente a init_index
                    init_index=(init_index+paso)&mask;
                }
                //Evaluamos el caso que no se consiguió el vértice
                if (!vex_found){
                    //Evaluamos para los siguientes beams
                    continue;
                }
                /*Procedemos a hallar los demás vértices del tripivot*/
                //Dependiendo del sector estamos más cerca del beam=0 o el beam=15
                k_beam=(n_beams-1)*(1.0-((sector>>1)&0x1));
                ///Hallamos los otros vértices
                get_tripivot(&v0,&v1,&Point_Cloud[v2*3],sector,i,k_beam);
                /*Debemos establecer el orden de los vértices, es decir horario o antihorario*/
                //Esto depende del sector
                if ((sector>>1)&0x1){
                    //definimos el sentido horario Ya que para los sectors 3 y 4
                    //El sentido de los vértices es distinto a los de los
                    //primeros sector. Entonces para seguir la jerarquía de los
                    //sentidos, cambiamos aqui
                    //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
                    //%%% Esto podría ser o o importante                        %
                    //%%% Capaz, se puede definir un sentido para un lado y otro%
                    //%%% para los otros sector ()sector3 y sector4)            %
                    //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
                    v_temp=v0;
                    v0=v1;
                    v1=v_temp;
                }
                /*Debemos verificar que los vértices no sean cero */
                //Analizamos el punto del vertice v0
                x_point=Point_Cloud[v0*3+0];
                y_point=Point_Cloud[v0*3+1];
                z_point=Point_Cloud[v0*3+2];
                if ((x_point==0)&&(y_point==0)&&(z_point==0)){
                    //Guardamos el valor de v1.//v_temp=v1;
                    get_tripivot(&v0,&v_temp,&Point_Cloud[v1*3],sector,i-1,k_beam);
                    if ((sector>>1)&0x1){
                        //Deseo el nuevo v0
                        v0=v_temp;
                    }
                    //Devolvemos el valor de v1.//v1=v_temp;
                }
                //Analizamos el punto del vertice v1
                x_point=Point_Cloud[v1*3+0];
                y_point=Point_Cloud[v1*3+1];
                z_point=Point_Cloud[v1*3+2];
                if ((x_point==0)&&(y_point==0)&&(z_point==0)){
                    get_tripivot(&v_temp,&v1,&Point_Cloud[v0*3],sector,i-1,k_beam);
                    if ((sector>>1)&0x1){
                        //Deseo el nuevo v1
                        v1=v_temp;
                    }
                }
                /*Debemos verificar concurrencia de vértices en los triángulos pívots con
                los últimos triangulos pivots de la Donut anterior con el mismo sector*/
                if (!(((sector>>1)&0x1)^(sector&0x1))){
                    //sector 1 y 4
                    if (v1==last_Tripivots[sector*3+1]){
                        v1=last_Tripivots[sector*3+2];
                    }
                }else{
                    //sector 2 y 3
                    if (v0==last_Tripivots[sector*3+0]){
                        v0=last_Tripivots[sector*3+2]; 
                    }
                }
                Tripivot[n_tripivot*3+0]=v0;
                Tripivot[n_tripivot*3+1]=v1;
                Tripivot[n_tripivot*3+2]=v2;
                n_tripivot++;
            }
            //Guardamos los triángulos hallados 
            for (unsigned int j= 0; j < n_tripivot; j++){
                TwoDF[n_Twotriangles*3+0]=Tripivot[j*3+0];
                TwoDF[n_Twotriangles*3+1]=Tripivot[j*3+1];
                TwoDF[n_Twotriangles*3+2]=Tripivot[j*3+2];
                n_Twotriangles++;
            }
            //Hallamos los 4 triangulos que limitan la zona del medio de cada sector,
            //para ello, guardamos el triángulo más cercano al medio
            Tripivot_middle[sector*3+0]=Tripivot[((n_tripivot-1)*(sector>>1))*3+0];
            Tripivot_middle[sector*3+1]=Tripivot[((n_tripivot-1)*(sector>>1))*3+1];
            Tripivot_middle[sector*3+2]=Tripivot[((n_tripivot-1)*(sector>>1))*3+2];
            if ((i==(n_donuts-1))&&(n_tripivot>1)){
                //Este caso ocurre en la última Donut, también hallar los 4 triangulos
                Tripivot_middle_particular[sector*3+0]=Tripivot[((n_tripivot-1)*(0x1^(sector>>1)))*3+0];
                Tripivot_middle_particular[sector*3+1]=Tripivot[((n_tripivot-1)*(0x1^(sector>>1)))*3+1];
                Tripivot_middle_particular[sector*3+2]=Tripivot[((n_tripivot-1)*(0x1^(sector>>1)))*3+2];
            }
            /**************************************FIN TRIPIVOT************************************/
            //Ahora con los triángulos pivot hallados, procedemos a realizar el llenado de las zonas.
            /**************************************************************************************/
            /**************************************************************************************/
            /**********                                                                ************/
            /**********                 TRIDONUTFILL && TWODONUTFILL                   ************/
            /**********                                                                ************/
            /**************************************************************************************/
            /**************************************************************************************/
            for (unsigned int j = 0; j < n_tripivot-1; j++){
                //Debemos clasifiicar la zona a llenar, por eso debemos obtener el punto siguiente 
                //del vertice v2
                v_temp=Tripivot[j*3+2];                
                //Particularmente, en los sector 0 y 2 hallamos el punto siguiente pero en los otros
                //sectores 1 y 3, hallamos el 17mo punto consecuente. Al analizar si este punto es
                //nulo o no, podemos definir si es una zona tipo escalera o rampa(ver pagina 19 de 
                //la presentación)
                x_point=Point_Cloud[(v_temp+1+n_beams*((sector)&0x1))*3+0];
                y_point=Point_Cloud[(v_temp+1+n_beams*((sector)&0x1))*3+1];
                z_point=Point_Cloud[(v_temp+1+n_beams*((sector)&0x1))*3+2];
                //Evaluamos la condición y definimos la forma de la zona
                if ((x_point==0)||(y_point==0)||(z_point==0)){
                    //El punto es cero, ha sido suprimido
                    escalera=sector&0x1;
                }else{
                    escalera=!(sector&0x1);
                }
                //Definimos el tipo
                tipo=0x1^((sector&0x1)^escalera);
                //Luego de definir la forma y el tipo. Podemos definir el valor de pasoL1 para llegar al v_fin
                pasoL0=n_beams*(1.0-2*(escalera^tipo));
                //Ahora debemos definir el vertice que estará en la esquina y su lugar puede estar en el beam 
                //actual o en el beam siguiente o en el beam del siguiente/anterior azimut
                v_corner=Tripivot[(j+tipo)*3+2]+(1.0-2*tipo)+pasoL0*(0x1^escalera);
                //definimos los límites
                v_init=Tripivot[(j+tipo)*3+1-escalera];
                v_fin=Tripivot[(j+(0x1^tipo))*3+1-(0x1^escalera)];
                v_corner_fin=Tripivot[(j+(0x1^tipo))*3+2];
                pasoL1=(1.0-2*(((sector>>1)&0x1)^tipo))*pasoL0;
                //Para verificar si es de TWO o TRI llenado, se tiene que analizar la condición de
                //que los vértices pertenezcan a una misma donut. Algo particular que podemos 
                //destacar es que si es TriDonutFill, necesariamente uno de los vértices pertenece
                //a la Donuut Referencial. 
                //Para saber si NO es TriDonutFill basta verificar el MSB del vértice, en este caso
                //si n_point_perDonut=(2>>14), habría que analizar el bit 14 y ver que este sea distinto
                //de cero, lo que es lo mismo decir que el vértice no pertene a la Donut referencial
                //Por lo tanto, para verificar si es TwoDonutFill basta cumplir alguna de estas 
                //condiciones:
                //  -Si al hacer el bitshift, ambos vertices tienen un valor distinto de cero
                //  -Si al hacer el bitshift, ambos vertices tienenun  valor igual a cero.
                //Lo anterior se puede hacer con bitwise and: vex&(~mask), para ambos casos
                if ((((~mask)&v_init)==(v_fin&(~mask)))||(((v_fin&(~mask))!=0)&&((v_init&(~mask))!=0))){
                    /* TwoDonutFill*/
                    //Creamos el primer triangulo
                    v0=Tripivot[(j+tipo)*3+2];
                    v1=v_init;
                    v2=v_corner;
                    //Guardamos el triángulo
                    TwoDF[n_Twotriangles*3+0]=v0;
                    TwoDF[n_Twotriangles*3+1]=v1;
                    TwoDF[n_Twotriangles*3+2]=v2;
                    n_Twotriangles++;
                    //Creamos la superficie formada por los dos triángulos pivot
                    side2sideFill(v_corner,v_init,v_corner_fin,v_fin,pasoL0,pasoL1,TwoDF,&n_Twotriangles);
                }else{
                    /* TriDonutFill*/
                    //Definimos un nuevo triángulo, que será el del medio
                    //primero tenemos que partir de un vertice para hallar al
                    //v2_mid. Este vertice de partida será siempre del T_actual
                    //para los sector 1 y 2 y para los sector3y4 será el T_next
                    //notar que se está hallando el tripivot de la Donut
                    //anterior (i-1). Pero para "angulos particulares" el tripivot
                    //pertenece a la siguiente Donut!!! (con suerte, este no es el caso)
                    v2_mid=Tripivot[(j+((sector>>1)&0x1))*3+0];//---->es indiferente si es el vertice 0 o 1, están en la misma recta(?)
                    paso_mid=pasoL1*(1.0-2.0*(tipo^((sector>>1)&0x1)));
                    //no hace falta hacer mask ya que siempre es con la Donut
                    //anterior y los Trifill ocurren a partir de la donut 2, es
                    //decir i>=3. Además, el tema de concatenación ocurre más en 
                    //la donut referencial
                    x_point=Point_Cloud[(v2_mid+paso_mid)*3+0];
                    y_point=Point_Cloud[(v2_mid+paso_mid)*3+1];
                    z_point=Point_Cloud[(v2_mid+paso_mid)*3+2];
                    freepoints_mid=0;
                    while ((x_point!=0)||(y_point!=0)||(z_point!=0)){
                        v2_mid+=paso_mid;
                        freepoints_mid++;//realizado el conteo de cuantos puntos libres
                        x_point=Point_Cloud[(v2_mid+paso_mid)*3+0];
                        y_point=Point_Cloud[(v2_mid+paso_mid)*3+1];
                        z_point=Point_Cloud[(v2_mid+paso_mid)*3+2];    
                    }
                    //Hallamos el triangulo pivot Repetimos el código de líneas arriba
                    //------------------TRIPIVOT MIDDLE----------------//
                    k_beam=(n_beams-1)*(1.0-((sector>>1)&0x1));
                    //realizamos la operación
                    get_tripivot(&v0_mid,&v1_mid,&Point_Cloud[v2_mid*3],sector,i-1,k_beam);
                    if ((sector>>1)&0x1){
                        //definimos el sentido horario Ya que para los sectors 3 y 4
                        //El sentido de los vértices es distinto a los de los
                        //primeros sector. Entonces para seguir la jerarquía de los
                        //sentidos, cambiamos aqui
                        //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
                        //%%% Esto podría ser o o importante                        %
                        //%%% Capaz, se puede definir un sentido para un lado y otro%
                        //%%% para los otros sector ()sector3 y sector4)                  %
                        //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
                        v_temp=v0_mid;
                        v0_mid=v1_mid;
                        v1_mid=v_temp;
                    }
                    T_mid[0]=v0_mid;
                    T_mid[1]=v1_mid;
                    T_mid[2]=v2_mid;
                    /*------------Fin TRIPIVOT MIDDLE-------------*/
                    /*----------Primer llenado----------*/
                    //Definimos los v_lim Para hacer el primer llenado
                    a=(sector>>1)&0x1;
                    b=sector&0x1;
                    c=escalera;
                    //v0_lim será el vertice limite para el v_corner
                    v0_lim=Tripivot[(j+(tipo^0x1))*3+2];
                    //v1_lim será el vertice limite para el v_init
                    index= 2 - (a^(0x1^(b^c))) - (0x1^((a^b)|c));
                    v1_lim=T_mid[index];
                    //procedemos a realizar el primer llenado
                    //Creamos el primer triangulo
                    v0=Tripivot[(j+tipo)*3+2];
                    v1=v_init;
                    v2=v_corner;
                    TriDF[n_Tritriangles*3+0]=v0;
                    TriDF[n_Tritriangles*3+1]=v1;
                    TriDF[n_Tritriangles*3+2]=v2;
                    n_Tritriangles++;
                    side2sideFill(v2,v1,v0_lim,v1_lim,pasoL0,pasoL1,TriDF,&n_Tritriangles);
                    /*----------Fin Primer llenado----------*/
                    /*----------Segundo llenado----------*/
                    index=((sector>>1)&0x1)^((sector&0x1)^escalera);
                    index= 2- index- (escalera&&(0x1^((sector&0x1)^((sector>>1)&0x1))));
                    //Creamos el triangulo de transicion
                    v0=v1_lim;
                    v1=T_mid[index];//Definimos el nuevo v_init
                    v2=v_corner_fin;
                    TriDF[n_Tritriangles*3+0]=v0;
                    TriDF[n_Tritriangles*3+1]=v1;
                    TriDF[n_Tritriangles*3+2]=v2;
                    n_Tritriangles++;
                    side2sideFill(v2,v1,v_corner_fin,v_fin,pasoL0,pasoL1,TriDF,&n_Tritriangles);
                    /*----------Fin Segundo llenado----------*/
                }
            }
            /*--------------------FIN FILL sector--------------------*/
            //la siguiente variable guardará los tripivots pasados para verificar que 
            //no exista concurrencia de tripivots con la siguiente Donut
            if (i!=n_donuts){
                //hacemos este condicional ya que para la última Donut no
                //necesitamos realizar esto
                last_Tripivots[sector*3+0]=Tripivot[(0x1^((sector>>1)&0x1))*(n_tripivot-1)*3+0];
                last_Tripivots[sector*3+1]=Tripivot[(0x1^((sector>>1)&0x1))*(n_tripivot-1)*3+1];
                last_Tripivots[sector*3+2]=Tripivot[(0x1^((sector>>1)&0x1))*(n_tripivot-1)*3+2];
            }            
        }
        /*Middle Fill*/
        //sector1 con sector2
        vleft_init=Tripivot_middle[2];
        vrigth_init=Tripivot_middle[1];
        vleft_fin=Tripivot_middle[3+2];
        vrigth_fin=Tripivot_middle[3+0];
        pasoLeft=n_beams;
        pasoRigth=pasoLeft;
        side2sideFill(vleft_init,vrigth_init,vleft_fin,vrigth_fin,pasoLeft,pasoRigth,MidDF,&n_Midtriangles);
        //sector3 con sector4
        vleft_init=Tripivot_middle[3*3+2];
        vrigth_init=Tripivot_middle[3*3+1];
        vleft_fin=Tripivot_middle[2*3+2];
        vrigth_fin=Tripivot_middle[2*3+0];
        pasoLeft=-n_beams;
        pasoRigth=pasoLeft;
        side2sideFill(vleft_init,vrigth_init,vleft_fin,vrigth_fin,pasoLeft,pasoRigth,MidDF,&n_Midtriangles);
        if ((i==(n_donuts-1))&&(n_tripivot>1)){
            vrigth_init=Tripivot_middle_particular[0];
            vleft_init=Tripivot_middle_particular[2];
            vrigth_fin=Tripivot_middle_particular[3+1];
            vleft_fin=Tripivot_middle_particular[3+2];
            pasoRigth=-n_beams;
            pasoLeft=-pasoRigth;
            side2sideFill(vleft_init,vrigth_init,vleft_fin,vrigth_fin,pasoLeft,pasoRigth,MidDF,&n_Midtriangles);
            vrigth_init=Tripivot_middle_particular[3*3+0];
            vleft_init=Tripivot_middle_particular[3*3+2];
            vrigth_fin=Tripivot_middle_particular[2*3+1];
            vleft_fin=Tripivot_middle_particular[2*3+2];
            pasoRigth=n_beams;
            pasoLeft=-pasoRigth;
            side2sideFill(vleft_init,vrigth_init,vleft_fin,vrigth_fin,pasoLeft,pasoRigth,MidDF,&n_Midtriangles);
        }
    }
}
/*----------------------------------------------------------------------------*/
/**
 * \brief Generate_surface. Genera la superficie
 * 
 * \param Point_Cloud es el puntero que tendrá los puntos de la nube de puntos 
 * de la data real  
 * 
 * \param T es el puntero donde se almacenará los vértices de los triángulos de 
 * la data real
 * 
 * \param pointer_n_triangles. Es un puntero que contendrá el valor de la cantidad 
 * de triángulos generados. Este valor definirá el límite de líneas para los 
 * archivos de reconstrucción
 * 
 * \return None
 */
void Generate_surface(double* Point_Cloud,unsigned int* T,unsigned int *pointer_n_triangles){
    double* Sphere_Cloud;
    Sphere_Cloud = (double*)malloc(n_total_points * 3 *sizeof(double));
    unsigned int *T_temp;
    T_temp=(unsigned int*)malloc(n_total_triangles * 3 *sizeof(unsigned int));
    Generate_sphere(Sphere_Cloud);
    Supress_redundant_data(Sphere_Cloud);
    One_Donut_Fill(Sphere_Cloud,T_temp);
    TwoandTri_Donut_Fill(Sphere_Cloud,&T_temp[OneDonutFill_triangles*3],&T_temp[(OneDonutFill_triangles+TwoDonutFill_triangles)*3],&T_temp[(OneDonutFill_triangles+TwoDonutFill_triangles+TriDonutFill_triangles)*3]);    
    /*Procedemos a chequear que los vertices son no nulos*/
    unsigned int temp_vex,n_triangles=0;
    double xp,yp,zp;
    for (unsigned int i = 0; i < n_total_triangles; i++){
        //Analizamos el punto del vertice v0
        temp_vex=T_temp[i*3];
        xp=Point_Cloud[temp_vex*3+0];
        yp=Point_Cloud[temp_vex*3+1];
        zp=Point_Cloud[temp_vex*3+2];
        if ((xp!=0)&&(yp!=0)&&(zp!=0)){
            //analizamos el punto del vertice v1
            temp_vex=T_temp[i*3+1];
            xp=Point_Cloud[temp_vex*3+0];
            yp=Point_Cloud[temp_vex*3+1];
            zp=Point_Cloud[temp_vex*3+2];
            if ((xp!=0)&&(yp!=0)&&(zp!=0)){
                //analizamos el punto del vertice v2
                temp_vex=T_temp[i*3+2];
                xp=Point_Cloud[temp_vex*3+0];
                yp=Point_Cloud[temp_vex*3+1];
                zp=Point_Cloud[temp_vex*3+2];
                if ((xp!=0)&&(yp!=0)&&(zp!=0)){
                    //Si todo lo anterior se cumple, guardamos el triángulo
                    T[n_triangles*3+2]=temp_vex;
                    T[n_triangles*3+1]=T_temp[i*3+1];
                    T[n_triangles*3]=T_temp[i*3];
                    n_triangles++;
                }
            }
        }
    }
    pointer_n_triangles[0]=n_triangles; 
    free(Sphere_Cloud);
    free(T_temp);   
}
/*----------------------------------------------------------------------------*/
/**
 * \brief multmatrix. function multiplica matrices para ser usado dentro del GPU 
 */
__device__ void mult_matrix_dev(double* A,unsigned int m,unsigned int n,double*B,unsigned int l,double*C) {
    for (unsigned int i = 0; i < l;i++) {
        for (unsigned int j = 0; j < m; j++) {
            C[j*l+i]=0;
            for (unsigned int k = 0; k < n; k++) {
                C[j * l +i] += A[j*n+k]*B[k*l + i];
            }
        }
    }
}
__device__ void rot_x_axis_dev(double* XYZ_points,double angle ){
    double temp[3], rot_matrix[9] = { 1,0,0,0,cos(angle),-sin(angle),0,sin(angle),cos(angle) };
    temp[0]=XYZ_points[0];
    temp[1]=XYZ_points[1];
    temp[2]=XYZ_points[2];
    mult_matrix_dev(rot_matrix, 3, 3, temp, 1, XYZ_points);
}
/**
 * \brief GenerateSphereCUDA. genera esfera 
 */
__global__ void GenerateSphereCUDA(double* Point_Cloud){
    double beam_altitude_angles[n_beams]= {15.379*D180_MPI,13.236*D180_MPI,11.128*D180_MPI,9.03*D180_MPI,6.941*D180_MPI,4.878*D180_MPI,2.788*D180_MPI,0.705*D180_MPI,-1.454*D180_MPI,-3.448*D180_MPI,-5.518*D180_MPI,-7.601*D180_MPI,-9.697*D180_MPI,-11.789*D180_MPI,-13.914*D180_MPI,-16.062*D180_MPI};
    double beam_azimuth_angles[n_beams] = { -1.24*D180_MPI, -1.2145*D180_MPI, -1.1889*D180_MPI, -1.1634*D180_MPI, -1.1379*D180_MPI, -1.1123*D180_MPI, -1.0868*D180_MPI, -1.0613*D180_MPI, -1.0357*D180_MPI, -1.0102*D180_MPI, -0.98467*D180_MPI, -0.95913*D180_MPI, -0.9336*D180_MPI, -0.90807*D180_MPI, -0.88253*D180_MPI, -0.857*D180_MPI };
    //Obtenemos el ID del thread
    int thid = threadIdx.x + blockIdx.x * blockDim.x;
	//Generamos el primer azimuth 
    if (thid<n_beams){
        //ubicamos el punto del azimut referencial en el plano XZ
        Point_Cloud[3 * thid + 0] = Radius_sphere* cos(beam_altitude_angles[thid] - MPI_2); //x
        Point_Cloud[3 * thid + 1] = 0;                                         //y
        Point_Cloud[3 * thid + 2] = Radius_sphere * sin(beam_altitude_angles[thid] - MPI_2);//z
        //Realizamos la rotacion del punto con respecto al eje x debido al desfase
        rot_x_axis_dev(&Point_Cloud[3*thid],beam_azimuth_angles[thid]);
        /*Creamos los azimuts que inician en cada sector*/
        /*mirror points from quarter Donut*/
        Point_Cloud[(thid + n_AZBLK / 4 * n_beams)*3+0] = Point_Cloud[3 * thid + 0];
        Point_Cloud[(thid+ n_AZBLK / 4 * n_beams)*3+1] = Point_Cloud[3 * thid + 2];
        Point_Cloud[(thid + n_AZBLK / 4 * n_beams)*3+2] = -Point_Cloud[3 * thid + 1];
        /*mirror points from midle Donut*/
        Point_Cloud[(thid + n_AZBLK / 2 * n_beams)*3+0] = Point_Cloud[3 * thid + 0];
        Point_Cloud[(thid + n_AZBLK / 2 * n_beams)*3+1] = -Point_Cloud[3 * thid + 1];
        Point_Cloud[(thid + n_AZBLK / 2 * n_beams)*3+2] = -Point_Cloud[3 * thid + 2];
        /*mirror points from 3 quater Donut*/
        Point_Cloud[(thid + n_AZBLK * 3 / 4 * n_beams)*3 +0] = Point_Cloud[3 * thid + 0];
        Point_Cloud[(thid + n_AZBLK * 3 / 4 * n_beams)*3 +1] = -Point_Cloud[3 * thid + 2];
        Point_Cloud[(thid + n_AZBLK * 3 / 4 * n_beams)*3 +2] = Point_Cloud[3 * thid + 1];
    }
    __syncthreads();
    /*Definimos la matrix de rotación para los azimuth*/
    /*Procedemos a realizar el barrido de cada sector para obtener la donnut referencial*/
    double temp[3];
    if(thid>0){
        double rot_matrix[9] = { 1,0,0,0,cos(angle_between_azimuths*thid),-sin(angle_between_azimuths*thid),0,sin(angle_between_azimuths*thid),cos(angle_between_azimuths*thid) };
        double XYZ[3];
        for (int j = 0; j < n_beams;j++) {
            //Obtain point from referencial azimuth
            XYZ[0] = Point_Cloud[j*3 + 0];
            XYZ[1] = Point_Cloud[j*3 + 1];
            XYZ[2] = Point_Cloud[j*3 + 2];
            //rotate that point
            mult_matrix_dev (rot_matrix,3,3,XYZ,1,temp);
            //Set the new azimuth
            Point_Cloud[(thid * n_beams + j)*3 + 0] = temp[0];
            Point_Cloud[(thid * n_beams + j)*3 + 1] = temp[1];
            Point_Cloud[(thid * n_beams + j)*3 + 2] = temp[2];
        }
    }
    __syncthreads();
    //Procedemos a rotar la Donut generada
    double rot_motor_matrix[9]={ cos(rot_angle),-sin(rot_angle),0,sin(rot_angle),cos(rot_angle) ,0,0,0,1 };
    for (unsigned int i = 1; i < n_donuts; i++) {
        //multiplicamos a todos los n_point_perdonut anteriores con la matriz de rotación
        for(int j=0;j<n_beams;j++){
            mult_matrix_dev(rot_motor_matrix, 3, 3, &Point_Cloud[((i - 1) * n_points_perDonut + thid*n_beams+j) * 3], 1, temp);
            Point_Cloud[(i * n_points_perDonut + thid*n_beams+j) * 3 + 0] = temp[0];
            Point_Cloud[(i * n_points_perDonut + thid*n_beams+j) * 3 + 1] = temp[1];
            Point_Cloud[(i * n_points_perDonut + thid*n_beams+j) * 3 + 2] = temp[2];        
        }
    }
    __syncthreads();
}

/**
 * \brief SupressOverlapCUDA. funcion en paralelo 
 */

__device__ double eq_line_dev(double m,double x,double xb,double yb) {
    double y= m*(x-xb)+yb;
    return y;
}
/**
 * \brief SupressOverlapCUDA. funcion en paralelo 
 */

__global__ void SupressOverlapCUDA(double* Point_Cloud){
    int thid = threadIdx.x + blockIdx.x * blockDim.x;
    //Declare temporary variables
    double x,y,y_temp,x1,y1,m;//xn,yn
    //Set vertical limits
    double X_L1=Point_Cloud[0],X_Ln=Point_Cloud[(n_beams-1)*3+0];
    //Array which contains lineś parameters
    double L[(n_donuts-2)*5];
    //Declare parameters from Donut_2 to Donut_n-1. Ya que para la supresión solo
    //se necesita los parametros de la Donut anterior y no necesitamos el de la
    //última Donut
    for (int i = 1; i < n_donuts-1; i++)
    {
        //Hallamos dos puntos de la recta proyectada en el plano XY
        x1=Point_Cloud[i*n_points_perDonut*3+0];
        y1=Point_Cloud[i*n_points_perDonut*3+1];
        x=Point_Cloud[i*n_points_perDonut*3+n_beams*3+0];
        y=Point_Cloud[i*n_points_perDonut*3+n_beams*3+1];
        //Calculamos la pendiente
        m=(y-y1)/(x-x1);
        //Guardamos los valores de un punto de la recta del beam_0
        L[(i-1)*5+0]=x1;
        L[(i-1)*5+1]=y1;
        //Guardamos los valores de un punto de la recta del beam_n
        L[(i-1)*5+2]=Point_Cloud[i*n_points_perDonut*3+n_beams*3-3];//xn
        L[(i-1)*5+3]=Point_Cloud[i*n_points_perDonut*3+n_beams*3-3+1];//yn
        //Guardamos el valor de la pendiente hallada
        L[(i-1)*5+4]=m;
    }
    //Supress redundant data only for Donut 2
    int i=1;
    for (unsigned int j = 0; j < n_beams; j++){
        //Calculamos si la coordenada del punto x
        x=Point_Cloud[(i*n_points_perDonut+thid*n_beams+j)*3+0];
        //Analizamos si se encuentra en la zona de supresión
        if (X_Ln<=x&&x<=X_L1){
            Point_Cloud[(i*n_points_perDonut+thid*n_beams+j)*3]=0;
            Point_Cloud[(i*n_points_perDonut+thid*n_beams+j)*3+1]=0;
            Point_Cloud[(i*n_points_perDonut+thid*n_beams+j)*3+2]=0;
            //points_deleted=points_deleted+1;
        }
    }
    __syncthreads();
    //Supress redundant for the rest of the Donuts
    //Creamos variable booleana para saber la zona del plano en donde se encuentra el punto
    bool left_side;
    unsigned int offset;
    for (unsigned int i = 2; i < n_donuts; i++){
        for (unsigned int j = 0; j < n_beams; j++){
            //Hallamos las coordenadas del punto a analizar
            offset=(i*n_points_perDonut+thid*n_beams+j)*3;
            x=Point_Cloud[offset];
            y=Point_Cloud[offset+1];
            //Evaluamos si se encuentra en la zona referencial
            if (X_Ln<=x){
                if(x<=X_L1){
                    Point_Cloud[offset]=0;
                    Point_Cloud[offset+1]=0;
                    Point_Cloud[offset+2]=0;
                    continue;
                }else{
                    //Se encuentra del lado derecho
                    left_side=false;
                }
            }else{
                //Se encuentra del lazo izquierdo
                left_side=true;
            }
            //Calculamos el valor de y_temp el cual limitará la zona
            y_temp=eq_line_dev(L[(i-2)*5+4],x,L[(i-2)*5+left_side*2],L[(i-2)*5+left_side*2+1]);
            //Le colocamos un signo negativo, o no, para poder realizar un único condicional para ambos casos
            y_temp=y_temp*(1.0-2*left_side);
            y=y*(1.0-2*left_side);
            //Evaluamos la condición de supresión
            if (y>=y_temp){
                //Eliminamos los puntos
                Point_Cloud[offset]=0;
                Point_Cloud[offset+1]=0;
                Point_Cloud[offset+2]=0;
                //points_deleted=points_deleted+1;
            }
        }   
    }
    __syncthreads();
}
/**
 * \brief OneDonutFillCUDA. First Mesh 
 */
__global__ void OneDonutFillCUDA(double* Point_Cloud,unsigned int* T,unsigned int* index_offset){
    int thid = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int v0,v1,v2;
    //Definimos los vértices
    //Realizamos la malla triangular para la Donut referencial
    for (unsigned int j = 0; j < n_beams-1; j++)
    {   
        v0=thid*n_beams+j;
        v2=v0+1;
        v1=(v0+n_beams+1)&mask;
        T[thid*(n_beams-1)*3*2+j*6]=v0;
        T[thid*(n_beams-1)*3*2+j*6+1]=v1;
        T[thid*(n_beams-1)*3*2+j*6+2]=v2;
        v2=v1;
        v1=v2-1;
        T[thid*(n_beams-1)*3*2+j*6+3]=v0;
        T[thid*(n_beams-1)*3*2+j*6+4]=v1;
        T[thid*(n_beams-1)*3*2+j*6+5]=v2;
    }
    __syncthreads();
    //En base a la malla referencial hallamos las demás superficies
    double xp,yp,zp;
    unsigned int count=0,offset,temp_vex,n_triangles_perThread=2*(n_beams-1);
    //creamos un Mesh array temporary
    unsigned int *T_per_thread;
    T_per_thread=(unsigned int*)malloc(5*n_triangles_perThread* 3 *sizeof(unsigned int));
    for (unsigned int i = 1; i < n_donuts; i++){
        //Analizamos cada vertice del tríangulo
        for (unsigned int j = 0; j < n_triangles_perThread; j++){
             //Analizamos el punto del vertice v0
            offset=(thid*n_triangles_perThread+j)*3;
            temp_vex=(T[offset]+i*n_points_perDonut);
            xp=Point_Cloud[temp_vex*3+0];
            yp=Point_Cloud[temp_vex*3+1];
            zp=Point_Cloud[temp_vex*3+2];
            if ((xp!=0)||(yp!=0)||(zp!=0)){
                //analizamos el punto del vertice v1
                temp_vex=(T[offset+1]+i*n_points_perDonut);
                xp=Point_Cloud[temp_vex*3+0];
                yp=Point_Cloud[temp_vex*3+1];
                zp=Point_Cloud[temp_vex*3+2];
                if ((xp!=0)||(yp!=0)||(zp!=0)){
                    //analizamos el punto del vertice v2
                    temp_vex=(T[offset+2]+i*n_points_perDonut);
                    xp=Point_Cloud[temp_vex*3+0];
                    yp=Point_Cloud[temp_vex*3+1];
                    zp=Point_Cloud[temp_vex*3+2];
                    if ((xp!=0)||(yp!=0)||(zp!=0)){
                        //Si todo lo anterior se cumple, guardamos el triángulo
                        T_per_thread[count*3+2]=temp_vex;
                        T_per_thread[count*3+1]=(T[offset+1]+i*n_points_perDonut);
                        T_per_thread[count*3]=(T[offset]+i*n_points_perDonut);
                        count++;
                    }
                }
            }
        }
    }
    index_offset[thid]=count;
    __syncthreads();
    unsigned int n_triangles_perDonut=n_AZBLK*2*(n_beams-1);
    //Ha este punto, cada hilo contiene una cierta cantidad de n triangulos que han de ser colocadas en el array original
    if(thid>0){
        offset=0;
        for(int z=0;z<thid;z++){
            offset+=index_offset[z];
        }
        offset*=3;
        //copy triangles 
        for(int z=0;z<count*3;z++){
            T[n_triangles_perDonut*3+offset+z]=T_per_thread[z];
        }
    }
    free(T_per_thread);
    __syncthreads();
}


__global__ void ODF_part1(double* Point_Cloud,unsigned int* T,unsigned int* T_temp,unsigned int* count_array){
    int thid = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int v0,v1,v2;
    //Definimos los vértices
    //Realizamos la malla triangular para la Donut referencial
    for (unsigned int j = 0; j < n_beams-1; j++)
    {   
        v0=thid*n_beams+j;
        v2=v0+1;
        v1=(v0+n_beams+1)&mask;
        T[thid*(n_beams-1)*3*2+j*6]=v0;
        T[thid*(n_beams-1)*3*2+j*6+1]=v1;
        T[thid*(n_beams-1)*3*2+j*6+2]=v2;
        v2=v1;
        v1=v2-1;
        T[thid*(n_beams-1)*3*2+j*6+3]=v0;
        T[thid*(n_beams-1)*3*2+j*6+4]=v1;
        T[thid*(n_beams-1)*3*2+j*6+5]=v2;
    }
    __syncthreads();
    //En base a la malla referencial hallamos las demás superficies
    double xp,yp,zp;
    unsigned int count=0,offset,temp_vex,n_triangles_perThreadandDonut=2*(n_beams-1);
    for (unsigned int i = 1; i < n_donuts; i++){
        //Analizamos cada vertice del tríangulo
        for (unsigned int j = 0; j < n_triangles_perThreadandDonut; j++){
             //Analizamos el punto del vertice v0
            offset=(thid*n_triangles_perThreadandDonut+j)*3;
            temp_vex=(T[offset]+i*n_points_perDonut);
            xp=Point_Cloud[temp_vex*3+0];
            yp=Point_Cloud[temp_vex*3+1];
            zp=Point_Cloud[temp_vex*3+2];
            if ((xp!=0)||(yp!=0)||(zp!=0)){
                //analizamos el punto del vertice v1
                temp_vex=(T[offset+1]+i*n_points_perDonut);
                xp=Point_Cloud[temp_vex*3+0];
                yp=Point_Cloud[temp_vex*3+1];
                zp=Point_Cloud[temp_vex*3+2];
                if ((xp!=0)||(yp!=0)||(zp!=0)){
                    //analizamos el punto del vertice v2
                    temp_vex=(T[offset+2]+i*n_points_perDonut);
                    xp=Point_Cloud[temp_vex*3+0];
                    yp=Point_Cloud[temp_vex*3+1];
                    zp=Point_Cloud[temp_vex*3+2];
                    if ((xp!=0)||(yp!=0)||(zp!=0)){
                        //Si todo lo anterior se cumple, guardamos el triángulo
                        T_temp[(n_donuts-1)*n_triangles_perThreadandDonut*3*thid+count*3+2]=temp_vex;
                        T_temp[(n_donuts-1)*n_triangles_perThreadandDonut*3*thid+count*3+1]=(T[offset+1]+i*n_points_perDonut);
                        T_temp[(n_donuts-1)*n_triangles_perThreadandDonut*3*thid+count*3]=(T[offset]+i*n_points_perDonut);
                        count++;
                    }
                }
            }
        }
    }
    count_array[thid]=count;
    __syncthreads();
}
#define ngpu 1024
#define NUM_BANKS 32
#define LOG_NUM_BANKS 5
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS)
__global__ void eScanGPU(unsigned int *g_odata, unsigned int *g_idata)
{
	__shared__ unsigned int temp[2*ngpu];// allocated on invocation
	int thid = threadIdx.x;
	int offset = 1,n=ngpu;
	
	int ai = thid;
	int bi = thid + (n/2);
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
	int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
	temp[ai + bankOffsetA] = g_idata[ai];
	temp[bi + bankOffsetB] = g_idata[bi];

//	temp[2*thid] = g_idata[2*thid]; // load input into shared memory
//	temp[2*thid+1] = g_idata[2*thid+1];
	
	for (int d = n>>1; d > 0; d >>= 1) // build sum in place up the tree
	{
		__syncthreads();
		if (thid < d)
		{
			int ai = offset*(2*thid+1)-1;
			int bi = offset*(2*thid+2)-1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);
//			int ai = offset*(2*thid+1)-1;
//			int bi = offset*(2*thid+2)-1;
			temp[bi] += temp[ai];
		}
		offset *= 2;
	}
	if (thid==0) { temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0; }
	//	if (thid == 0) { temp[n - 1] = 0; } // clear the last element
	for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (thid < d)
		{
			int ai = offset*(2*thid+1)-1;
			int bi = offset*(2*thid+2)-1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);
//			int ai = offset*(2*thid+1)-1;
//			int bi = offset*(2*thid+2)-1;

			unsigned int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();
	g_odata[ai] = temp[ai + bankOffsetA];
	g_odata[bi] = temp[bi + bankOffsetB];
//	g_odata[2*thid] = temp[2*thid]; // write results to device memory
//	g_odata[2*thid+1] = temp[2*thid+1];
}



 __global__ void ODF_part2(unsigned int* T,unsigned int* T_temp,unsigned int* count_array,unsigned int* index_offset){
    int thid = threadIdx.x + blockIdx.x * blockDim.x;   
    unsigned int count,offset,n_triangles_perDonut=n_AZBLK*2*(n_beams-1),n_triangles_perThreadandDonut=2*(n_beams-1);
    count=count_array[thid];
    //Ha este punto, cada hilo contiene una cierta cantidad de n triangulos que han de ser colocadas en el array original
    if(thid==0)
    offset=0;
    else
    offset=index_offset[thid]*3;
    //copy triangles 
    for(int z=0;z<count*3;z++){
        T[n_triangles_perDonut*3+offset+z]=T_temp[(n_donuts-1)*n_triangles_perThreadandDonut*3*thid+z];
    }
    __syncthreads();
}

void Generate_surfaceGPU(double* Point_Cloud,unsigned int* T,unsigned int *pointer_n_triangles){
    /////////////////////////////////////////////////////////////////////////////////////////
    ///////////////////////////////////      CUDA        ////////////////////////////////////
    /////////////////////////////////////////////////////////////////////////////////////////
    hipError_t cudaerr;
    double* Sphere_Cloud_dev;
    hipMalloc((void**)(&Sphere_Cloud_dev), sizeof(double) * n_total_points * 3);
    //1st step. GenerateSphere
    GenerateSphereCUDA<<<numBlocks, threadsPerBlock >>> (Sphere_Cloud_dev);
    hipDeviceSynchronize();
    //2nd step. Overlap removing
    SupressOverlapCUDA<<<numBlocks, threadsPerBlock >>> (Sphere_Cloud_dev);
    double* Sphere_Cloud;
    Sphere_Cloud = (double*)malloc(n_total_points * 3 *sizeof(double));
    cudaerr=hipMemcpy(Sphere_Cloud, Sphere_Cloud_dev, sizeof(double) *n_total_points * 3, hipMemcpyDeviceToHost);
    if (cudaerr != 0)	printf("ERROR copying to SphereCloud. CudaMalloc value=%i\n\r",cudaerr);
    unsigned int *OneDonutMesh_dev,*T_temp_dev,*count_array_dev,n_triangles_perDonut=n_AZBLK*2*(n_beams-1);
    hipMalloc((void**)(&OneDonutMesh_dev), sizeof(unsigned int) * OneDonutFill_triangles * 3);
    hipMalloc((void**)(&T_temp_dev), sizeof(unsigned int) * n_triangles_perDonut* (n_donuts-1) * 3);
    hipMalloc((void**)(&count_array_dev), sizeof(unsigned int) * threadsPerBlock);
    //3rd step. First part of the ODF
    ODF_part1<<<numBlocks, threadsPerBlock >>> (Sphere_Cloud_dev,OneDonutMesh_dev,T_temp_dev,count_array_dev);
    unsigned int *index_offset_array_dev;
    hipMalloc((void**)(&index_offset_array_dev), sizeof(unsigned int) * 1024);
    //4th step. eScan GPU
    hipDeviceSynchronize();
    eScanGPU<<<numBlocks, threadsPerBlock/2 >>> (index_offset_array_dev,count_array_dev);
    //5th step. last part ODF
    hipDeviceSynchronize();
    ODF_part2<<<numBlocks, threadsPerBlock >>> (OneDonutMesh_dev,T_temp_dev,count_array_dev,index_offset_array_dev);
    hipDeviceSynchronize();
    unsigned int *T_temp;
    T_temp=(unsigned int*)malloc(n_total_triangles * 3 *sizeof(unsigned int));
    cudaerr=hipMemcpy(T_temp,OneDonutMesh_dev, sizeof(unsigned int) *OneDonutFill_triangles * 3, hipMemcpyDeviceToHost);
    if (cudaerr != 0)	printf("ERROR copying to T_temp. CudaMalloc value=%i\n\r",cudaerr);
	/////////////////////////////////////////////////////////////////////////////////////////
    ///////////////////////////////////      CPU        ////////////////////////////////////
    /////////////////////////////////////////////////////////////////////////////////////////
    //6th step. last Fill
    TwoandTri_Donut_Fill(Sphere_Cloud,&T_temp[OneDonutFill_triangles*3],&T_temp[(OneDonutFill_triangles+TwoDonutFill_triangles)*3],&T_temp[(OneDonutFill_triangles+TwoDonutFill_triangles+TriDonutFill_triangles)*3]);
    unsigned int temp_vex,n_triangles=0;
    double xp,yp,zp;
    for (unsigned int i = 0; i < n_total_triangles; i++){
        //Analizamos el punto del vertice v0
        temp_vex=T_temp[i*3];
        xp=Point_Cloud[temp_vex*3+0];
        yp=Point_Cloud[temp_vex*3+1];
        zp=Point_Cloud[temp_vex*3+2];
        if ((xp!=0)&&(yp!=0)&&(zp!=0)){
            //analizamos el punto del vertice v1
            temp_vex=T_temp[i*3+1];
            xp=Point_Cloud[temp_vex*3+0];
            yp=Point_Cloud[temp_vex*3+1];
            zp=Point_Cloud[temp_vex*3+2];
            if ((xp!=0)&&(yp!=0)&&(zp!=0)){
                //analizamos el punto del vertice v2
                temp_vex=T_temp[i*3+2];
                xp=Point_Cloud[temp_vex*3+0];
                yp=Point_Cloud[temp_vex*3+1];
                zp=Point_Cloud[temp_vex*3+2];
                if ((xp!=0)&&(yp!=0)&&(zp!=0)){
                    //Si todo lo anterior se cumple, guardamos el triángulo
                    T[n_triangles*3+2]=temp_vex;
                    T[n_triangles*3+1]=T_temp[i*3+1];
                    T[n_triangles*3]=T_temp[i*3];
                    n_triangles++;
                }
            }
        }
    }
    pointer_n_triangles[0]=n_triangles; 
    hipFree(Sphere_Cloud_dev);
    hipFree(OneDonutMesh_dev);
    hipFree(T_temp_dev);
    hipFree(count_array_dev);
    hipFree(index_offset_array_dev);
    free(Sphere_Cloud);
    free(T_temp);
}



/******************************************************************/
/*************************       MAIN     *************************/
/******************************************************************/
#define TESTING 0
int main()
{
#if TESTING == 0
    #define iter 500.0
    /*Allocate memory*/
    double* Point_Cloud;
    Point_Cloud = (double*)malloc(n_total_points * 3 *sizeof(double));
    //Leemos del csv los datos reales
    FILE* archivo;
    archivo = fopen("../files/MinaData.csv", "r");
    char buffer[200];
    char* token;
    //Saltamos la primera línea
    fgets(buffer,sizeof(buffer),archivo);
    for (unsigned int i = 0; i < n_total_points; i++){
        fgets(buffer,sizeof(buffer),archivo);
        token = strtok(buffer,",");
        Point_Cloud[i*3+0]=atof(token);
        token = strtok(NULL,",");
        Point_Cloud[i*3+1]=atof(token);
        token = strtok(NULL,",\n");
        Point_Cloud[i*3+2]=atof(token);
    }    
    fclose(archivo);    
    
    /*****************************************************/
    /********************       CPU     ******************/
    /*****************************************************/
    unsigned int *T_cpu,n_triangles_real_data_cpu;
    T_cpu=(unsigned int*)malloc(n_total_triangles * 3 *sizeof(unsigned int));
    clock_t startCPU;
	clock_t finishCPU;
    printf ("/********************solo CPU*********************/:\n");
	startCPU = clock();
    for (int i=0;i<iter;i++){
        Generate_surface(Point_Cloud,T_cpu,&n_triangles_real_data_cpu);
    }
	finishCPU = clock();
	printf("numero de triangulos: %d\n",n_triangles_real_data_cpu);
    printf("CPU: %fms\n", ((double)(finishCPU - startCPU))/(double)CLOCKS_PER_SEC);
	
    /*****************************************************/
    /********************       GPU     ******************/
    /*****************************************************/
    unsigned int *T_gpu,n_triangles_real_data_gpu;
    T_gpu=(unsigned int*)malloc(n_total_triangles * 3 *sizeof(unsigned int));
    clock_t startGPU;
	clock_t finishGPU;
    printf ("/********************CPU y GPU*********************/:\n");
	startGPU = clock();
    for (int i=0;i<iter;i++){
        Generate_surfaceGPU(Point_Cloud,T_gpu,&n_triangles_real_data_gpu);
    }
	finishGPU = clock();
	printf("numero de triangulos: %d\n",n_triangles_real_data_gpu);
    printf("GPU: %fms\n", ((double)(finishGPU - startGPU))/(double)CLOCKS_PER_SEC);
	//creamos archivo para ver results
    archivo = fopen("../files/CUDAMesh.csv", "w+");
    fprintf(archivo, "V1, V2, V3\n");
    for (unsigned int i=0; i < n_triangles_real_data_gpu; i++) {
        fprintf(archivo,"%d, %d, %d\n", T_gpu[i*3+0], T_gpu[i * 3 + 1], T_gpu[i * 3 + 2]);
    }
    fclose(archivo);
    //------------------------------------------
	//----------Generate the DXF file-----------
	//------------------------------------------
	//Open the DXF file
    archivo = fopen("../files/CUDAMinaSurface.dxf", "w");
    //assert(archivo);
    //header
    fprintf(archivo, "0\nSECTION\n2\nENTITIES\n0\n");
	float x0,y0,z0,x1,y1,z1,x2,y2,z2;
	for (int i = 0; i < n_triangles_real_data_gpu; i++)
    {
        // get the coordinates of each point from the triangle
        x0 = Point_Cloud[T_gpu[i* 3+0]*3 + 0];
        y0 = Point_Cloud[T_gpu[i* 3+0]*3 + 1];
        z0 = Point_Cloud[T_gpu[i* 3+0]*3 + 2];
        
        x1 = Point_Cloud[T_gpu[i* 3+1]*3 + 0];
        y1 = Point_Cloud[T_gpu[i* 3+1]*3 + 1];
        z1 = Point_Cloud[T_gpu[i* 3+1]*3 + 2];
        
        x2 = Point_Cloud[T_gpu[i* 3+2]*3 + 0];
        y2 = Point_Cloud[T_gpu[i* 3+2]*3 + 1];
        z2 = Point_Cloud[T_gpu[i* 3+2]*3 + 2];
        //create new 3DFACE element
        fprintf(archivo, "3DFACE\n8\n1\n");
        fprintf(archivo, " 62\n %d\n", 142);//corresponding color of the autocad pallete
        fprintf(archivo, "10\n %.4f\n 20\n %.4f\n 30\n %.4f\n", x0, y0, z0);
        fprintf(archivo, "11\n %.4f\n 21\n %.4f\n 31\n %.4f\n", x1, y1, z1);
        fprintf(archivo, "12\n %.4f\n 22\n %.4f\n 32\n %.4f\n", x2, y2, z2);
        fprintf(archivo, "13\n %.4f\n 23\n %.4f\n 33\n %.4f\n", x2, y2, z2);
        fprintf(archivo, "0\n");
    }
    fprintf(archivo, "ENDSEC\n 0\nEOF\n");
    fclose(archivo);

    return 0;
#else
    #define iter 500.0
    /*************************************************************************************/
    /***********************************    CPU     **************************************/
    /*************************************************************************************/
    double* Point_Cloud;
    Point_Cloud = (double*)malloc(n_total_points * 3 *sizeof(double));
    unsigned int *T;
    T=(unsigned int*)malloc(n_total_triangles * 3 *sizeof(unsigned int));

    clock_t start,stop;
    double timeCPU=0;
    printf ("/********************solo CPU*********************/:\n");
    start=clock();
    for (int z=0;z<iter;z++)
    Generate_sphere(Point_Cloud);
    stop=clock();
    timeCPU+=((double)(stop - start))*1000.0/(double)CLOCKS_PER_SEC/iter;
    printf("GS time: %fms\n", ((double)(stop - start))*1000.0/(double)CLOCKS_PER_SEC/iter);
    
    start=clock();
    for (int z=0;z<iter;z++)
    Supress_redundant_data(Point_Cloud);
    stop=clock();
    timeCPU+=((double)(stop - start))*1000.0/(double)CLOCKS_PER_SEC/iter;
    printf("OR time: %fms\n", ((double)(stop - start))*1000.0/(double)CLOCKS_PER_SEC/iter);
    
    start=clock();
    for (int z=0;z<iter;z++)
    One_Donut_Fill(Point_Cloud,T);
    stop=clock();
    timeCPU+=((double)(stop - start))*1000.0/(double)CLOCKS_PER_SEC/iter;
    printf("ODF time: %fms\n", ((double)(stop - start))*1000.0/(double)CLOCKS_PER_SEC/iter);
    
    start=clock();
    for (int z=0;z<iter;z++)
    TwoandTri_Donut_Fill(Point_Cloud,&T[OneDonutFill_triangles*3],&T[(OneDonutFill_triangles+TwoDonutFill_triangles)*3],&T[(OneDonutFill_triangles+TwoDonutFill_triangles+TriDonutFill_triangles)*3]);    
    stop=clock();
    timeCPU+=((double)(stop - start))*1000.0/(double)CLOCKS_PER_SEC/iter;
    printf("LastFill time: %fms\n", ((double)(stop - start))*1000.0/(double)CLOCKS_PER_SEC/iter);
    printf("total time: %fms\n",timeCPU);
        
    free(Point_Cloud);
    free(T);
    /*************************************************************************************/
    /***********************************    GPU     **************************************/
    /*************************************************************************************/    
    double* Point_Cloud_dev,*Point_Cloud_gpu;
    cudaMalloc((void**)(&Point_Cloud_dev), sizeof(double) * n_total_points * 3);
    Point_Cloud_gpu = (double*)malloc(n_total_points * 3 *sizeof(double));

    cudaError_t cudaerr;
    cudaEvent_t start_gpu, stop_gpu;
    float timeGPU,totalGPU=0;
    cudaEventCreate(&start_gpu);
    cudaEventCreate(&stop_gpu);

    printf ("/*********************CPU y GPU********************/\n");
    cudaEventRecord(start_gpu);
    for (int z = 0; z < iter; z++)
    GenerateSphereCUDA<<<numBlocks, threadsPerBlock >>> (Point_Cloud_dev);
    cudaEventRecord(stop_gpu);
    cudaEventSynchronize(stop_gpu);
	cudaEventElapsedTime(&timeGPU, start_gpu, stop_gpu);
    printf("GS time:  %fms\n\r", timeGPU / iter);
    totalGPU+=timeGPU/iter;
    cudaDeviceSynchronize();

    cudaEventRecord(start_gpu);
    for (int z = 0; z < iter; z++)
    SupressOverlapCUDA<<<numBlocks, threadsPerBlock >>> (Point_Cloud_dev);
    cudaEventRecord(stop_gpu);
    cudaEventSynchronize(stop_gpu);
	cudaEventElapsedTime(&timeGPU, start_gpu, stop_gpu);
    cudaerr=cudaMemcpy(Point_Cloud_gpu, Point_Cloud_dev, sizeof(double) *n_total_points * 3, cudaMemcpyDeviceToHost);
    printf("OR time:  %fms\n\r", timeGPU / iter);
    totalGPU+=timeGPU/iter;

    unsigned int *T_gpu,*OneMesh_dev,*OneMesh_temp_dev,n_triangles_perDonut=n_AZBLK*2*(n_beams-1);
    unsigned int *count_array_dev,*index_offset_array_dev;
    
    cudaMalloc((void**)(&OneMesh_dev), sizeof(unsigned int) * n_total_triangles * 3);
    cudaMalloc((void**)(&OneMesh_temp_dev), sizeof(unsigned int) * n_triangles_perDonut* (n_donuts-1) * 3);
    cudaMalloc((void**)(&count_array_dev), sizeof(unsigned int) * threadsPerBlock);
    cudaMalloc((void**)(&index_offset_array_dev), sizeof(unsigned int) * 1024);
    T_gpu=(unsigned int*)malloc(n_total_triangles * 3 *sizeof(unsigned int));

    //3rd step. First part of the ODF
    cudaDeviceSynchronize();
    cudaEventRecord(start_gpu);
    for (int z = 0; z < iter; z++){
        ODF_part1<<<numBlocks, threadsPerBlock >>> (Point_Cloud_dev,OneMesh_dev,OneMesh_temp_dev,count_array_dev);
        cudaDeviceSynchronize();
        //4th step. eScan GPU
        eScanGPU<<<numBlocks, threadsPerBlock/2 >>> (index_offset_array_dev,count_array_dev);
        cudaDeviceSynchronize();
        //5th step. last part ODF
        ODF_part2<<<numBlocks, threadsPerBlock >>> (OneMesh_dev,OneMesh_temp_dev,count_array_dev,index_offset_array_dev);
        cudaDeviceSynchronize();
    
    }
    cudaEventRecord(stop_gpu);
    cudaEventSynchronize(stop_gpu);
	cudaEventElapsedTime(&timeGPU, start_gpu, stop_gpu);
    cudaerr=cudaMemcpy(T_gpu,OneMesh_dev, sizeof(unsigned int) *OneDonutFill_triangles * 3, cudaMemcpyDeviceToHost);
    if (cudaerr != 0)	printf("ERROR copying to T_gpu. CudaMalloc value=%i\n\r",cudaerr);
    printf("ODF time:  %fms\n\r", timeGPU / iter);
    totalGPU+=timeGPU/iter;

    start=clock();
    for (int z=0;z<iter;z++)
    TwoandTri_Donut_Fill(Point_Cloud_gpu,&T_gpu[OneDonutFill_triangles*3],&T_gpu[(OneDonutFill_triangles+TwoDonutFill_triangles)*3],&T_gpu[(OneDonutFill_triangles+TwoDonutFill_triangles+TriDonutFill_triangles)*3]);    
    stop=clock();
    totalGPU+=((double)(stop - start))*1000.0/(double)CLOCKS_PER_SEC/iter;
    printf("LastFill time: %fms\n", ((double)(stop - start))*1000.0/(double)CLOCKS_PER_SEC/iter);
    
    printf("total time: %fms\n",totalGPU);
    //Finalmente, liberamos el resto de memoria
    cudaFree(Point_Cloud_dev);
    cudaFree(OneMesh_dev);
    cudaFree(OneMesh_temp_dev);
    cudaFree(count_array_dev);
    cudaFree(index_offset_array_dev);
    free(Point_Cloud_gpu);
    free(T_gpu); 
    return 0;
#endif
}