﻿
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdint.h>
#include <stdlib.h>
#include "math.h"
#include "time.h"
#ifndef MPI//definimos el valor de PI
#define MPI 3.14159265358979323846
#endif
//#include "mkl.h"
/*----------------------------------------------------------------------------*/
/**
 * Definimos los parámetros del LIDAR
 * Se puede usar typedef para juntar los parámetros del LIDAR
 */
/*Cantidad de rayos por azimuth*/
#define n_beams 16
/*Cantidad de azimuths por Donut*/
#define n_AZBLK 1024
/*Los ángulos por defecto de fabrica del LIDAR*/
double beam_altitude_angles[n_beams]= {15.379,13.236,11.128,9.03,6.941,4.878,2.788,0.705,-1.454,-3.448,-5.518,-7.601,-9.697,-11.789,-13.914,-16.062};
double beam_azimuth_angles[n_beams] = { -1.24, -1.2145, -1.1889, -1.1634, -1.1379, -1.1123, -1.0868, -1.0613, -1.0357, -1.0102, -0.98467, -0.95913, -0.9336, -0.90807, -0.88253, -0.857 };
/*Cantidad puntos de la nube*/
#define n_points_perDonut (n_AZBLK*n_beams)
/*Angulo entre azimuths*/
#define angle_between_azimuths (-2*MPI/n_AZBLK)
/*Ángulo de rotacion del motor*/
#define rot_angle (-33.53706667 * MPI / 180)
/*Cantidad de Donuts en función del ángulo de rotación*/
unsigned char n_donuts = (unsigned char)ceil(-MPI/rot_angle);
#define n_total_points (n_points_perDonut*n_donuts)
/*----------------------------------------------------------------------------*/
/**
 * Funciones de conversión
 */
void rad2deg(double *value) {
    *value = (*value) * 180 / MPI;
}
void deg2rad(double *value) {
    value[0] = value[0] * MPI / 180;
}
void mult_matrix(double* A,unsigned int m,unsigned int n,double*B,unsigned int l,double*C) {
    for (unsigned int i = 0; i < l;i++) {
        for (unsigned int j = 0; j < m; j++) {
            C[j*l+i]=0;
            for (unsigned int k = 0; k < n; k++) {
                C[j * l +i] += A[j*n+k]*B[k*l + i];
            }
        }
    }
}
/*----------------------------------------------------------------------------*/
/** 
 * Funciones de matrices de rotación
 */
void rot_x_axis(double* XYZ_points,double angle ){
    double temp[3], rot_matrix[9] = { 1,0,0,0,cos(angle),-sin(angle),0,sin(angle),cos(angle) };
    memcpy(temp, XYZ_points, 3*sizeof(double));
    mult_matrix(rot_matrix, 3, 3, temp, 1, XYZ_points);
}
void rot_y_axis(double* XYZ_points, double angle) {
    double temp[3], rot_matrix[9] = { cos(angle),0,sin(angle),0,1,0,-sin(angle),0,cos(angle) };
    memcpy(temp, XYZ_points, 3 * sizeof(double));
    mult_matrix(rot_matrix, 3, 3, temp, 1, XYZ_points);
}
void rot_z_axis(double* XYZ_points, double angle) {
    double temp[3], rot_matrix[9] = { cos(angle),-sin(angle),0,sin(angle),cos(angle) ,0,0,0,1};
    memcpy(temp, XYZ_points, 3 * sizeof(double));
    mult_matrix(rot_matrix, 3, 3, temp, 1, XYZ_points);
}
/*----------------------------------------------------------------------------*/
/**
 * \brief Generate synthetic sphere. La generación se divide en 3 estapas. La 
 * 1ra etapa consta en la generación de un azimut referencial; para ello, se 
 * define que el azimuth referencial será aquel que se genera en la vertical
 * inferior, luego, ubicamos cada punto en el plano XZ y luego le realizamos la
 * rotación en el eje X debido al desfase de rayos en cada azimut. En la 2da 
 * etapa se realiza el barrido del azimut referencial para obtener la Donut 
 * refernecial, y en la 3era etapa se realiza la rotación a la Donut referencial
 * para obtener la esfera modelada.
 * 
 * \param Point_Cloud es el puntero que tendrá los puntos de la nube de puntos
 * de la esfera sintética modelada
 * 
 * \return None
 */
#define Radius_sphere 1.0
void Generate_sphere(double* Point_Cloud) {
    double x,y,z,R= Radius_sphere;
    for (int i = 0; i < n_beams; i++) {
        /*ubicamos el punto del azimuth*/
        /*x = r * cos(beam_altitude_angles[i] - mpi / 2);
        y = 0;
        z = r * sin(beam_altitude_angles[i] - mpi / 2);




        temp = rot_x_axis(beam_azimuth_angles[i]) * [x, y, z]';
            point_cloud(j, 1) = temp(1);
        point_cloud(j, 2) = temp(2);
        point_cloud(j, 3) = temp(3);*/
    }


}
/*----------------------------------------------------------------------------*/

int main()
{  
    /** 
    * Nuestro sistema de referencia será : Eje Z será el eje de giro del motor.
    *                                      Eje X será el eje de la Donut referencial
    */
    /*Creamos la matrix de rotación del eje del motor (eje Z)*/
    double rot_matrix[9] = { cos(rot_angle),-sin(rot_angle),0,sin(rot_angle),cos(rot_angle) ,0,0,0,1 };
    /*Allocate memory*/
    double* Point_Cloud;
    Point_Cloud= (double*)malloc(n_total_points*sizeof(double));











    //------------------------------//
    /*Testing function*/
    double A[12] = { 1,2,3,4,5,6,7,8,9,10,11,12 };
    double B[12] = { 10,7,4,1,11,8,5,2,12,9,6,3};
    double C[16];
    mult_matrix(A, 1, 3, B, 4, C);
    for (int z=0; z < 1; z++) {
        for (int w=0; w < 4; w++) {
            printf("%.3f\t", C[z*4+w]);
        }
        printf("\n");
    }
    //-----------------------------// 
    /*Testing rot axis*/
    double temp[3] = { 2,4,5 };
    printf("Antes:\n%.3f\t%.3f\t%.3f\n", temp[0], temp[1], temp[2]);
    rot_x_axis(temp,MPI);
    printf("Despues:\n%.3f\t%.3f\t%.3f\n", temp[0], temp[1], temp[2]);
    //----------------------------//
    /*Testing n_donuts value*/
    printf("Hola mundo %d %f\n", n_donuts,beam_altitude_angles[2]);
    /* for (int i=0;i<n_beams; i++) {
        printf("%f\n", beam_altitude_angles[i]);
    }*/
    return 0;
}
