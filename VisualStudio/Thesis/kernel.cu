﻿
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdint.h>
#include <stdlib.h>
#include "math.h"
#include "time.h"
#ifndef MPI//definimos el valor de PI
#define MPI 3.14159265358979323846
#endif
//#include "mkl.h"
/*----------------------------------------------------------------------------*/
/**
 * Definimos los parámetros del LIDAR
 * Se puede usar typedef para juntar los parámetros del LIDAR
 */
/*Cantidad de rayos por azimuth*/
#define n_beams 16
/*Cantidad de azimuths por Donut*/
#define n_AZBLK 1024
/*Los ángulos por defecto de fabrica del LIDAR*/
double beam_altitude_angles[n_beams]= {15.379,13.236,11.128,9.03,6.941,4.878,2.788,0.705,-1.454,-3.448,-5.518,-7.601,-9.697,-11.789,-13.914,-16.062};
double beam_azimuth_angles[n_beams] = { -1.24, -1.2145, -1.1889, -1.1634, -1.1379, -1.1123, -1.0868, -1.0613, -1.0357, -1.0102, -0.98467, -0.95913, -0.9336, -0.90807, -0.88253, -0.857 };
/*Cantidad puntos de la nube*/
#define n_points_perDonut (unsigned int)(n_AZBLK*n_beams)
/*Angulo entre azimuths*/
#define angle_between_azimuths (-2*MPI/n_AZBLK)
/*Ángulo de rotacion del motor*/
#define rot_angle (-33.53706667 * MPI / 180)
/*Creamos la matrix de rotación del eje del motor (eje Z)*/
double rot_motor_matrix[9] ={ cos(rot_angle),-sin(rot_angle),0,sin(rot_angle),cos(rot_angle) ,0,0,0,1 };
/*Cantidad de Donuts en función del ángulo de rotación*/
#define n_donuts (unsigned int)ceil(-MPI/rot_angle)
#define n_total_points (unsigned int)(n_donuts*n_points_perDonut)
/*----------------------------------------------------------------------------*/
/**
 * Funciones de conversión
 */
void rad2deg(double *value) {
    *value = (*value) * 180 / MPI;
}
void deg2rad(double *value) {
    value[0] = value[0] * MPI / 180;
}
void mult_matrix(double* A,unsigned int m,unsigned int n,double*B,unsigned int l,double*C) {
    for (unsigned int i = 0; i < l;i++) {
        for (unsigned int j = 0; j < m; j++) {
            C[j*l+i]=0;
            for (unsigned int k = 0; k < n; k++) {
                C[j * l +i] += A[j*n+k]*B[k*l + i];
            }
        }
    }
}
/*----------------------------------------------------------------------------*/
/** 
 * Funciones de matrices de rotación
 */
void rot_x_axis(double* XYZ_points,double angle ){
    double temp[3], rot_matrix[9] = { 1,0,0,0,cos(angle),-sin(angle),0,sin(angle),cos(angle) };
    memcpy(temp, XYZ_points, 3*sizeof(double));
    mult_matrix(rot_matrix, 3, 3, temp, 1, XYZ_points);
}
void rot_y_axis(double* XYZ_points, double angle) {
    double temp[3], rot_matrix[9] = { cos(angle),0,sin(angle),0,1,0,-sin(angle),0,cos(angle) };
    memcpy(temp, XYZ_points, 3 * sizeof(double));
    mult_matrix(rot_matrix, 3, 3, temp, 1, XYZ_points);
}
void rot_z_axis(double* XYZ_points, double angle) {
    double temp[3], rot_matrix[9] = { cos(angle),-sin(angle),0,sin(angle),cos(angle) ,0,0,0,1};
    memcpy(temp, XYZ_points, 3 * sizeof(double));
    mult_matrix(rot_matrix, 3, 3, temp, 1, XYZ_points);
}
/*----------------------------------------------------------------------------*/
/**
 * \brief Generate synthetic sphere. La generación se divide en 3 estapas. La 
 * 1ra etapa consta en la generación de un azimut referencial; para ello, se 
 * define que el azimuth referencial será aquel que se genera en la vertical
 * inferior, luego, ubicamos cada punto en el plano XZ y luego le realizamos la
 * rotación en el eje X debido al desfase de rayos en cada azimut. En la 2da 
 * etapa se realiza el barrido del azimut referencial para obtener la Donut 
 * refernecial, y en la 3era etapa se realiza la rotación a la Donut referencial
 * para obtener la esfera modelada.
 * 
 * \param Point_Cloud es el puntero que tendrá los puntos de la nube de puntos
 * de la esfera sintética modelada
 * 
 * \return None
 */
#define Radius_sphere 1.0
void Generate_sphere(double* Point_Cloud) {
    double R= Radius_sphere;
    /*Generamos el azimuth refencial y los azimuts de cada sector*/
    for (int i = 0; i < n_beams; i++) {
        /*ubicamos el punto del azimut referencial*/
        Point_Cloud[3 * i + 0] = R* cos(beam_altitude_angles[i] - MPI / 2); //x
        Point_Cloud[3 * i + 1] = 0;                                         //y
        Point_Cloud[3 * i + 2] = R * sin(beam_altitude_angles[i] - MPI / 2);//z
        /*Realizamos la rotacion del punto con respecto al eje x debido al desfase*/
        rot_x_axis(&Point_Cloud[3*i],beam_azimuth_angles[i]);
        /*Creamos los azimuts que inician en cada sector*/
        /*mirror points from quarter Donut*/
        Point_Cloud[(i + n_AZBLK / 4 * n_beams)*3+0] = Point_Cloud[3 * i + 0];
        Point_Cloud[(i + n_AZBLK / 4 * n_beams)*3+1] = Point_Cloud[3 * i + 2];
        Point_Cloud[(i + n_AZBLK / 4 * n_beams)*3+2] = -Point_Cloud[3 * i + 1];
        /*mirror points from midle Donut*/
        Point_Cloud[(i + n_AZBLK / 2 * n_beams)*3+0] = Point_Cloud[3 * i + 0];
        Point_Cloud[(i + n_AZBLK / 2 * n_beams)*3+1] = -Point_Cloud[3 * i + 1];
        Point_Cloud[(i + n_AZBLK / 2 * n_beams)*3+2] = -Point_Cloud[3 * i + 2];
        /*mirror points from 3 quater Donut*/
        Point_Cloud[(i + n_AZBLK * 3 / 4 * n_beams)*3 +0] = Point_Cloud[3 * i + 0];
        Point_Cloud[(i + n_AZBLK * 3 / 4 * n_beams)*3 +1] = -Point_Cloud[3 * i + 2];
        Point_Cloud[(i + n_AZBLK * 3 / 4 * n_beams)*3 +2] = Point_Cloud[3 * i + 1];

    }
    /*Definimos la matrix de rotación para los azimuth*/
    double rot_matrix[9] = { 1,0,0,0,cos(angle_between_azimuths),-sin(angle_between_azimuths),0,sin(angle_between_azimuths),cos(angle_between_azimuths) };
    /*Procedemos a realizar el barrido de cada sector para obtener la donnut referencial*/
    double XYZ[3],temp[9];
    for (int i = 1; i < n_AZBLK / 4; i++) {
        for (int j = 0; j < n_beams;j++) {
            /*Calculate previous point*/
            XYZ[0] = Point_Cloud[((i - 1) * n_beams + j)*3 + 0];
            XYZ[1] = Point_Cloud[((i - 1) * n_beams + j)*3 + 1];
            XYZ[2] = Point_Cloud[((i - 1) * n_beams + j)*3 + 2];
            /*rotate that point*/
            mult_matrix (rot_matrix,3,3,XYZ,1,temp);
            /*Set the new azimuth*/
            Point_Cloud[(i * n_beams + j)*3 + 0] = temp[0];
            Point_Cloud[(i * n_beams + j)*3 + 1] = temp[1];
            Point_Cloud[(i * n_beams + j)*3 + 2] = temp[2];
            /*mirror from quarter Donunt*/
            Point_Cloud[((i + n_AZBLK / 4) * n_beams + j)*3 + 0] = temp[0];
            Point_Cloud[((i + n_AZBLK / 4) * n_beams + j)*3 + 1] = temp[2];
            Point_Cloud[((i + n_AZBLK / 4) * n_beams + j)*3 + 2] = -temp[1];
            /*7mirror points from midle Donut*/
            Point_Cloud[((i + n_AZBLK / 2) * n_beams + j)*3+0] = temp[0];
            Point_Cloud[((i + n_AZBLK / 2) * n_beams + j)*3+1] = -temp[1];
            Point_Cloud[((i + n_AZBLK / 2) * n_beams + j)*3+2] = -temp[2];
            /*mirror points from midle Donut*/
            Point_Cloud[((i + n_AZBLK * 3 / 4) * n_beams + j)*3 + 0] = temp[0];
            Point_Cloud[((i + n_AZBLK * 3 / 4) * n_beams + j)*3 + 1] = -temp[2];
            Point_Cloud[((i + n_AZBLK * 3 / 4) * n_beams + j)*3 + 2] = temp[1];
        }
    }
    /*Rotamos la Donut referencial*/
    for (unsigned int i = 1; i < n_donuts; i++) {
        /*multiplicamos a todos los n_point_perdonut anteriores con la matriz de rotación*/
        for (unsigned int j = 0; j < n_points_perDonut; j++) {
            mult_matrix(rot_motor_matrix, 3, 3, &Point_Cloud[((i - 1) * n_points_perDonut + j) * 3], 1, temp);
            Point_Cloud[(i * n_points_perDonut + j) * 3 + 0] = temp[0];
            Point_Cloud[(i * n_points_perDonut + j) * 3 + 1] = temp[1];
            Point_Cloud[(i * n_points_perDonut + j) * 3 + 2] = temp[2];
;        }
    }
}
/*----------------------------------------------------------------------------*/

int main()
{
    /**
    * Nuestro sistema de referencia será : Eje Z será el eje de giro del motor.
    *                                      Eje X será el eje de la Donut referencial
    */
    /*Allocate memory*/
    double* Point_Cloud;
    Point_Cloud = (double*)malloc(n_total_points * 3 *sizeof(double));
    Generate_sphere(Point_Cloud);
    /*Escribimos la data obtenida en un archivo csv*/

    FILE* archivo;
    archivo = fopen("Sphere_cloud.csv", "w+");
    fprintf(archivo, "X, Y, Z\n");
    for (unsigned int i=0; i < n_total_points; i++) {
        fprintf(archivo,"%.4f, %.4f, %.4f\n", Point_Cloud[i*3+0], Point_Cloud[i * 3 + 1], Point_Cloud[i * 3 + 2]);
    }
    fclose(archivo);



    //------------------------------//
    /*Testing function*/
    double A[12] = { 1,2,3,4,5,6,7,8,9,10,11,12 };
    double B[12] = { 10,7,4,1,11,8,5,2,12,9,6,3};
    double C[16];
    mult_matrix(A, 1, 3, B, 4, C);
    for (int z=0; z < 1; z++) {
        for (int w=0; w < 4; w++) {
            printf("%.3f\t", C[z*4+w]);
        }
        printf("\n");
    }
    //-----------------------------// 
    /*Testing rot axis*/
    double temp[3] = { 2,4,5 };
    printf("Antes:\n%.3f\t%.3f\t%.3f\n", temp[0], temp[1], temp[2]);
    rot_x_axis(temp,MPI);
    printf("Despues:\n%.3f\t%.3f\t%.3f\n", temp[0], temp[1], temp[2]);
    //----------------------------//
    /*Testing n_donuts value*/
    printf("Hola mundo %d %f\n", n_donuts,beam_altitude_angles[2]);
    /* for (int i=0;i<n_beams; i++) {
        printf("%f\n", beam_altitude_angles[i]);
    }*/
    return 0;
}
