﻿
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include "math.h"
#ifndef MPI//definimos el valor de PI
#define MPI 3.14159265358979323846
#endif
//--------------------------------------------//
//-----Definimos los parámetros del LIDAR-----//
//--------------------------------------------//
//Cantidad de rayos por azimuth
#define n_beams 16
//Cantidad de azimuths por Donut
#define n_AZBLK 1024
//Los ángulos por defecto de fabrica del LIDAR
double beam_altitude_angles[n_beams]= {15.379,13.236,11.128,9.03,6.941,4.878,2.788,0.705,-1.454,-3.448,-5.518,-7.601,-9.697,-11.789,-13.914,-16.062};
double beam_azimuth_angles[n_beams] = { -1.24, -1.2145, -1.1889, -1.1634, -1.1379, -1.1123, -1.0868, -1.0613, -1.0357, -1.0102, -0.98467, -0.95913, -0.9336, -0.90807, -0.88253, -0.857 };
//Cantidad puntos de la nube
#define n_points (n_AZBLK*n_beams)
//Angulo entre azimuths
#define angle_between_azimuths (-2*MPI/n_AZBLK)
//Ángulo de rotacion del motor
#define rot_angle (-33.53706667 * MPI / 180);
//Cantidad de Donuts en función del ángulo de rotación
unsigned char n_donuts = (unsigned char)(-180 + beam_altitude_angles[0] - beam_altitude_angles[15])*MPI/180/rot_angle;

//--------------------------------------------//
//-----     Funciones de conversión      -----//
//--------------------------------------------//
void rad2deg(double *value) {
    *value = (*value) * 180 / MPI;
}
void deg2rad(double *value) {
    value[0] = value[0] * MPI / 180;
}

//--------------------------------------------//
//----- Funciones de matrices de rotación-----//
//--------------------------------------------//
void rot_matrix() {
}

void rot_x_axis() {
}

void rot_y_axis() {
}

void rot_z_axis() {
}
int main()
{  
     
    printf("Hola mundo %f %f\n",MPI,beam_altitude_angles[2]);
    return 0;
}
